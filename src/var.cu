#include "hip/hip_runtime.h"
#include "var.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include "precision.cuh"

dfloat res = 1.0;
int mesh = static_cast<int>(std::round(150 * res));

int nx = mesh;
int ny = mesh;
int nz = mesh;

// fluid velocity set
#ifdef FD3Q19
    int fpoints = 19;
#elif defined(FD3Q27)
    int fpoints = 27;
#endif

// phase velocity set
#ifdef PD3Q15
    int gpoints = 15;
#elif defined(PD3Q19)
    int gpoints = 19;
#elif defined(PD3Q27)
    int gpoints = 27;
#endif

dfloat tau = 0.505;
dfloat cssq = 1.0 / 3.0;
dfloat omega = 1.0 / tau;
dfloat sharp_c = 0.15 * 3.0;
dfloat sigma = 0.1;

dfloat *d_f, *d_g, *d_w, *d_w_g, *d_cix, *d_ciy, *d_ciz;
dfloat *d_normx, *d_normy, *d_normz, *d_indicator, *d_mod_grad;
dfloat *d_curvature, *d_ffx, *d_ffy, *d_ffz;
dfloat *d_ux, *d_uy, *d_uz, *d_pxx, *d_pyy, *d_pzz;
dfloat *d_pxy, *d_pxz, *d_pyz, *d_rho, *d_phi;
dfloat *d_fneq;
dfloat *d_grad_fix, *d_grad_fiy, *d_grad_fiz, *d_uu;

dfloat *h_pxx = (dfloat *)malloc(nx * ny * nz * sizeof(dfloat));
dfloat *h_pyy = (dfloat *)malloc(nx * ny * nz * sizeof(dfloat));
dfloat *h_pzz = (dfloat *)malloc(nx * ny * nz * sizeof(dfloat));
dfloat *h_pxy = (dfloat *)malloc(nx * ny * nz * sizeof(dfloat));
dfloat *h_pxz = (dfloat *)malloc(nx * ny * nz * sizeof(dfloat));
dfloat *h_pyz = (dfloat *)malloc(nx * ny * nz * sizeof(dfloat));

#ifdef FD3Q19
    const dfloat cix[19] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0 };
    const dfloat ciy[19] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 0, 0, 1, -1, -1, 1, 0, 0, 1, -1 };
    const dfloat ciz[19] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, -1, 1, -1, 1 };
#elif defined(FD3Q27)
    const dfloat cix[27] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 1, -1, 1, -1 };
    const dfloat ciy[27] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 0, 0, 1, -1, -1, 1, 0, 0, 1, -1, 1, -1, -1, 1, 1, -1, -1, 1 };
    const dfloat ciz[27] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, -1, 1, -1, 1, 1, -1, -1, 1, 1, 1, -1, -1 };
#endif

void initializeVars() {
    size_t size = nx * ny * nz * sizeof(dfloat);            
    size_t f_size = nx * ny * nz * fpoints * sizeof(dfloat); 
    size_t g_size = nx * ny * nz * gpoints * sizeof(dfloat); 
    size_t vs_size = fpoints * sizeof(dfloat);
    size_t pf_size = gpoints * sizeof(dfloat);
    size_t single_size = sizeof(dfloat);

    auto IDX3D = [&](int i, int j, int k) {
        return ((i) + nx * ((j) + ny * (k)));
    };

    for (int k = 0; k < nz; ++k) {
        for (int j = 0; j < ny; ++j) {
            for (int i = 0; i < nx; ++i) {
                h_pxx[IDX3D(i,j,k)] = 1.0;
                h_pyy[IDX3D(i,j,k)] = 1.0;
                h_pzz[IDX3D(i,j,k)] = 1.0;
                h_pxy[IDX3D(i,j,k)] = 1.0;
                h_pxz[IDX3D(i,j,k)] = 1.0;
                h_pyz[IDX3D(i,j,k)] = 1.0;
            }
        }
    }

    hipMalloc((void **)&d_rho, size);
    hipMalloc((void **)&d_phi, size);
    hipMalloc((void **)&d_ux, size);
    hipMalloc((void **)&d_uy, size);
    hipMalloc((void **)&d_uz, size);
    hipMalloc((void **)&d_normx, size);
    hipMalloc((void **)&d_normy, size);
    hipMalloc((void **)&d_normz, size);
    hipMalloc((void **)&d_curvature, size);
    hipMalloc((void **)&d_indicator, size);
    hipMalloc((void **)&d_ffx, size);
    hipMalloc((void **)&d_ffy, size);
    hipMalloc((void **)&d_ffz, size);
    hipMalloc((void **)&d_mod_grad, size);
    hipMalloc((void **)&d_pxx, size);
    hipMalloc((void **)&d_pyy, size);
    hipMalloc((void **)&d_pzz, size);
    hipMalloc((void **)&d_pxy, size);
    hipMalloc((void **)&d_pxz, size);
    hipMalloc((void **)&d_pyz, size);

    hipMalloc((void **)&d_f, f_size);
    hipMalloc((void **)&d_g, g_size);
    hipMalloc((void **)&d_w, vs_size);
    hipMalloc((void **)&d_w_g, pf_size);
    hipMalloc((void **)&d_cix, vs_size);
    hipMalloc((void **)&d_ciy, vs_size);
    hipMalloc((void **)&d_ciz, vs_size);
    hipMalloc((void **)&d_fneq, vs_size);

    hipMalloc((void **)&d_grad_fix, single_size);
    hipMalloc((void **)&d_grad_fiy, single_size);
    hipMalloc((void **)&d_grad_fiz, single_size);
    hipMalloc((void **)&d_uu, single_size);

    hipMemset(d_ux, 0.0, size);
    hipMemset(d_uy, 0.0, size);
    hipMemset(d_uz, 0.0, size);
    hipMemset(d_normx, 0.0, size);
    hipMemset(d_normy, 0.0, size);
    hipMemset(d_normz, 0.0, size);
    hipMemset(d_curvature, 0.0, size);
    hipMemset(d_indicator, 0.0, size);
    hipMemset(d_ffx, 0.0, size);
    hipMemset(d_ffy, 0.0, size);
    hipMemset(d_ffz, 0.0, size);
    hipMemset(d_mod_grad, 0.0, size);
    hipMemset(d_fneq, 0.0, vs_size);

    hipMemcpy(d_pxx, h_pxx, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pyy, h_pyy, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pzz, h_pzz, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pxy, h_pxy, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pxz, h_pxz, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pyz, h_pyz, size, hipMemcpyHostToDevice);
    hipMemcpy(d_cix, cix, vs_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ciy, ciy, vs_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ciz, ciz, vs_size, hipMemcpyHostToDevice);

    free(h_pxx);
    free(h_pyy);
    free(h_pzz);
    free(h_pxy);
    free(h_pxz);
    free(h_pyz);
}

