#include "kernels.cuh"
#include "auxFunctions.cuh"
#include "var.cuh"
#include "errorDef.cuh"
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>
#include <iomanip>

#include "precision.cuh"

int main(int argc, char* argv[]) {
    if (argc < 5) {
        cerr << "Erro: Uso: " << argv[0] << " F<fluid velocity set> P<phase field velocity set> <id> <save_binary>" << endl;
        return 1;
    }
    string fluid_model = argv[1];
    string phase_model = argv[2];
    string id = argv[3];
    bool save_binary = stoi(argv[4]);
    bool debug_mode = (argc > 5) ? stoi(argv[5]) : 0; 

    string base_dir;   
    #ifdef _WIN32
        base_dir = "..\\";
    #else
        base_dir = "../";
    #endif
    string model_dir = base_dir + "bin/" + fluid_model + "_" + phase_model + "/";
    string sim_dir = model_dir + id + "/";
    string matlab_dir = base_dir + "matlabFiles/" + fluid_model + "_" + phase_model + "/" + id + "/";

    if (!debug_mode) { 
        if (save_binary) {
            #ifdef _WIN32
                string mkdir_command = "mkdir \"" + sim_dir + "\"";
            #else
                string mkdir_command = "mkdir -p \"" + sim_dir + "\"";
            #endif
            int ret = system(mkdir_command.c_str());
            (void)ret;
        } else {
            #ifdef _WIN32
                string mkdir_command = "mkdir \"" + matlab_dir + "\"";
            #else
                string mkdir_command = "mkdir -p \"" + matlab_dir + "\"";
            #endif
            int ret = system(mkdir_command.c_str());
            (void)ret;
        }
    }

    // ========================= //
    int stamp = 1, nsteps = 10;
    // ========================= //
    initializeVars();

    if (!debug_mode && save_binary) {
        string info_file = sim_dir + id + "_info.txt";
        generateSimulationInfoFile(info_file, nx, ny, nz, stamp, nsteps, tau, id, fluid_model);
    }

    vector<dfloat> f(nx * ny * nz * fpoints, 0.0);
    vector<dfloat> g(nx * ny * nz * gpoints, 0.0);
    vector<dfloat> phi(nx * ny * nz, 0.0);
    vector<dfloat> rho(nx * ny * nz, 1.0); 

    // =========================== FLUID WEIGHTS =========================== //
    const vector<dfloat> w = {
        1.0 / 3.0, 
        1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0,
        1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0
    };
    // ===================================================================== //

    // ======================== PHASE FIELD WEIGHTS ======================== //
    const vector<dfloat> w_g = {
        2.0 / 9.0, 
        1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0,
        1.0 / 72.0, 1.0 / 72.0, 1.0 / 72.0, 1.0 / 72.0, 1.0 / 72.0, 1.0 / 72.0, 1.0 / 72.0, 1.0 / 72.0
    };
    // ===================================================================== //

    computeInitialCPU(phi, rho, w, w_g, f, g, nx, ny, nz, fpoints, gpoints, res);
    checkCudaErrors(hipMemcpy(d_f, f.data(), nx * ny * nz * fpoints * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_g, g.data(), nx * ny * nz * gpoints * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_phi, phi.data(), nx * ny * nz * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rho, rho.data(), nx * ny * nz * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_w, w.data(), fpoints * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_w_g, w_g.data(), gpoints * sizeof(dfloat), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

    vector<dfloat> phi_host(nx * ny * nz);

    for (int t = 0; t < nsteps; ++t) {

        cout << "Passo " << t << " de " << nsteps << " iniciado..." << endl;

        phiCalc<<<numBlocks, threadsPerBlock>>>(
            d_phi, d_g, 
            nx, ny, nz
        );
        hipDeviceSynchronize();
        
        gradCalc<<<numBlocks, threadsPerBlock>>>(
            d_phi, d_w, d_cix,
            d_grad_fix, d_grad_fiy, d_grad_fiz,
            d_ciy, d_ciz, fpoints,
            nx, ny, nz
        );
        hipDeviceSynchronize();
        
        normCalc<<<numBlocks, threadsPerBlock>>>(
            d_grad_fix, d_grad_fiy, d_grad_fiz,
            d_mod_grad, d_normx, d_normy, d_normz,
            d_indicator, nx, ny, nz
        );
        hipDeviceSynchronize();
        
        curvatureCalc<<<numBlocks, threadsPerBlock>>>(
            d_curvature, d_indicator, d_w,
            d_cix, d_ciy, d_ciz,
            d_normx, d_normy, d_normz,
            d_ffx, d_ffy, d_ffz, sigma,
            fpoints, nx, ny, nz
        );
        hipDeviceSynchronize();
        
        forceCalc<<<numBlocks, threadsPerBlock>>>(
            d_ffx, d_ffy, d_ffz,
            sigma, d_curvature, d_indicator,
            d_normx, d_normy, d_normz,
            nx, ny, nz
        );
        hipDeviceSynchronize();
        
        macroCalc<<<numBlocks, threadsPerBlock>>>(
            d_ux, d_uy, d_uz, d_f,
            d_ffx, d_ffy, d_ffz, d_rho,
            nx, ny, nz
        );
        hipDeviceSynchronize();
        
        uuCalc<<<numBlocks, threadsPerBlock>>>(
            d_ux, d_uy, d_uz, d_uu,
            cssq, nx, ny, nz
        );
        hipDeviceSynchronize();
        
        rhoCalc<<<numBlocks, threadsPerBlock>>>(
            d_rho, d_f, nx, ny, nz
        );
        hipDeviceSynchronize();
        
        momentiCalc<<<numBlocks, threadsPerBlock>>>(
            d_ux, d_uy, d_uz, d_w,
            d_cix, d_ciy, d_ciz,
            d_ffx, d_ffy, d_ffz,
            d_uu, d_rho, d_fneq,
            d_f, cssq, nx, ny, nz, 
            fpoints
        );
        hipDeviceSynchronize();
        
        tensorCalc<<<numBlocks, threadsPerBlock>>>(
            d_pxx, d_pyy, d_pzz,
            d_pxy, d_pxz, d_pyz,
            nx, ny, nz, d_fneq
        );
        hipDeviceSynchronize();

        uuCalc<<<numBlocks, threadsPerBlock>>>(
            d_ux, d_uy, d_uz, d_uu,
            cssq, nx, ny, nz
        );
        hipDeviceSynchronize();
        
        fCalc<<<numBlocks, threadsPerBlock>>>(
            d_ux, d_uy, d_uz,
            d_cix, d_ciy, d_ciz,
            d_w, d_rho, d_uu,
            d_ffx, d_ffy, d_ffz,
            d_pxx, d_pyy, d_pzz,
            d_pxy, d_pxz, d_pyz,
            d_f, omega, cssq,
            fpoints, nx, ny, nz
        );
        hipDeviceSynchronize();
        
        gCalc<<<numBlocks, threadsPerBlock>>>(
            d_ux, d_uy, d_uz,
            d_cix, d_ciy, d_ciz,
            d_w_g, d_phi, d_g,
            d_normx, d_normy, d_normz,
            cssq, gpoints, nx, ny, nz,
            sharp_c
        );
        hipDeviceSynchronize();
        
        streamingCalc<<<numBlocks, threadsPerBlock>>>(
            d_g, d_cix, d_ciy, d_ciz,
            nx, ny, nz, gpoints
        );
        hipDeviceSynchronize();
        
        fgBoundary<<<numBlocks, threadsPerBlock>>>(
            d_f, d_g, d_rho, d_phi, d_w, d_w_g,
            d_cix, d_ciy, d_ciz,
            fpoints, gpoints, nx, ny, nz
        );
        hipDeviceSynchronize();
        
        boundaryConditions<<<numBlocks, threadsPerBlock>>>(
            d_phi, nx, ny, nz
        );
        hipDeviceSynchronize();

        if (!debug_mode && t % stamp == 0) {
            checkCudaErrors(hipMemcpy(phi_host.data(), d_phi, nx * ny * nz * sizeof(dfloat), hipMemcpyDeviceToHost));
            
            if (save_binary) {
                ostringstream filename_phi_bin;
                filename_phi_bin << sim_dir << id << "_phi" << setw(6) << setfill('0') << t << ".bin";
                ofstream file_phi_bin(filename_phi_bin.str(), ios::binary);
                file_phi_bin.write(reinterpret_cast<const char*>(phi_host.data()), phi_host.size() * sizeof(dfloat));
                file_phi_bin.close();
            } else {
                ostringstream filename_phi_txt;
                filename_phi_txt << matlab_dir << id << "_phi" << t << ".txt";
                ofstream file_phi_txt(filename_phi_txt.str());
                if (file_phi_txt.is_open()) {
                    for (int z = 0; z < nz; ++z) {
                        for (int y = 0; y < ny; ++y) {
                            for (int x = 0; x < nx; ++x) {
                                int index = x + nx * (y + ny * z);
                                file_phi_txt << phi_host[index] << " ";
                            }
                            file_phi_txt << "\n";
                        }
                        file_phi_txt << "\n";
                    }
                    file_phi_txt.close();
                }
            }

            cout << "Passo " << t << ": Dados salvos em " << (save_binary ? sim_dir : matlab_dir) << endl;
        }
        
    }

    dfloat *pointers[] = {d_f, d_g, d_phi, d_rho, d_w, d_w_g, d_cix, d_ciy, d_ciz, 
                          d_mod_grad, d_normx, d_normy, d_normz, d_indicator,
                          d_curvature, d_ffx, d_ffy, d_ffz, d_ux, d_uy, d_uz,
                          d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, d_fneq,
                          d_grad_fix, d_grad_fiy, d_grad_fiz, d_uu
                        };
    freeMemory(pointers, 32);  

    return 0;
}
