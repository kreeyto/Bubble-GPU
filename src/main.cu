#include "kernels.cuh"
#include "utils.cuh"
#include "var.cuh"
#include "errorDef.cuh"
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>
#include <iomanip>

int main() {
    initializeVars();

    int stamp = 1, nsteps = 50;
    std::vector<float> phi_host(nx * ny * nz, 0.0f);
    std::string output_dir = "../bin/simulation/000/";
    std::string info_file = "../bin/simulation/000/000_info.txt";
    generateSimulationInfoFile(info_file, nx, ny, nz, stamp, nsteps, tau);

    std::vector<float> f(nx * ny * nz * fpoints, 0.0f);
    std::vector<float> g(nx * ny * nz * gpoints, 0.0f);
    std::vector<float> phi(nx * ny * nz, 0.0f);
    std::vector<float> rho(nx * ny * nz, 1.0f); 
    const std::vector<float> w = {
        1.0f / 3.0f, 
        1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f,
        1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f
    };
    const std::vector<float> w_g = {
        2.0f / 9.0f, 
        1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f,
        1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f
    };
    computeInitialCPU(phi, rho, w, w_g, f, g, nx, ny, nz, fpoints, gpoints, res);

    checkCudaErrors(hipMemcpy(d_f, f.data(), nx * ny * nz * fpoints * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_g, g.data(), nx * ny * nz * gpoints * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_phi, phi.data(), nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rho, rho.data(), nx * ny * nz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_w, w.data(), fpoints * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_w_g, w_g.data(), gpoints * sizeof(float), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

    for (int t = 0; t < nsteps; ++t) {

        std::cout << "Passo " << t << " de " << nsteps << " iniciado..." << std::endl;

        phiCalc<<<numBlocks, threadsPerBlock>>> (
            d_phi, d_g, gpoints, nx, ny, nz
        );
        getLastCudaError("Erro no kernel phiCalc");
        hipDeviceSynchronize();

        gradCalc<<<numBlocks, threadsPerBlock>>> (
            d_phi, d_mod_grad, d_normx, d_normy, d_normz, 
            d_indicator, d_w, d_cix, d_ciy, d_ciz, 
            fpoints, nx, ny, nz
        );
        getLastCudaError("Erro no kernel gradCalc");
        hipDeviceSynchronize();

        curvatureCalc<<<numBlocks, threadsPerBlock>>> (
            d_curvature, d_indicator, d_w,
            d_cix, d_ciy, d_ciz,
            d_normx, d_normy, d_normz, 
            d_ffx, d_ffy, d_ffz, sigma,
            fpoints, nx, ny, nz
        );
        getLastCudaError("Erro no kernel curvatureCalc");
        hipDeviceSynchronize();

        momentiCalc<<<numBlocks, threadsPerBlock>>> (
            d_ux, d_uy, d_uz, d_rho,
            d_ffx, d_ffy, d_ffz, d_w, d_f,
            d_cix, d_ciy, d_ciz, 
            d_pxx, d_pyy, d_pzz,
            d_pxy, d_pxz, d_pyz,
            cssq, nx, ny, nz,
            fpoints, d_fneq
        );
        getLastCudaError("Erro no kernel momentiCalc");
        hipDeviceSynchronize();

        collisionCalc<<<numBlocks, threadsPerBlock>>> (
            d_ux, d_uy, d_uz, d_w, d_w_g,
            d_cix, d_ciy, d_ciz,
            d_normx, d_normy, d_normz,
            d_ffx, d_ffy, d_ffz,
            d_rho, d_phi, d_f, d_g, 
            d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, 
            cssq, omega, sharp_c, fpoints, gpoints,
            nx, ny, nz
        );
        getLastCudaError("Erro no kernel collisionCalc");
        hipDeviceSynchronize();

        streamingCalc<<<numBlocks, threadsPerBlock>>> (
            d_g, d_cix, d_ciy, d_ciz, nx, ny, nz, gpoints
        );
        getLastCudaError("Erro no kernel streamingCalc");
        hipDeviceSynchronize();

        boundaryConditions<<<numBlocks, threadsPerBlock>>> (
            d_f, d_g, d_rho, d_phi, d_w, d_w_g,
            d_cix, d_ciy, d_ciz, fpoints, gpoints, nx, ny, nz
        );
        getLastCudaError("Erro no kernel boundaryConditions");
        hipDeviceSynchronize();

        if (t % stamp == 0) {

            std::ostringstream filename_phi;
            
            checkCudaErrors(hipMemcpy(phi_host.data(), d_phi, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
            filename_phi << output_dir << "000_phi" << std::setw(6) << std::setfill('0') << t << ".bin";
            std::ofstream file_phi(filename_phi.str(), std::ios::binary);
            file_phi.write(reinterpret_cast<const char*>(phi_host.data()), phi_host.size() * sizeof(float));
            file_phi.close();

            std::cout << "Passo " << t << ": Dados salvos em " << output_dir << std::endl;
        }
        
    }

    float *pointers[] = {d_f, d_g, d_phi, d_rho, d_w, d_w_g, d_cix, d_ciy, d_ciz, 
                     d_mod_grad, d_normx, d_normy, d_normz, d_indicator,
                     d_curvature, d_ffx, d_ffy, d_ffz, d_ux, d_uy, d_uz,
                     d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, d_fneq
                     };
    freeMemory(pointers, 28);  

    return 0;
}
