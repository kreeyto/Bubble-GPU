#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "var.cuh"
#include <math.h>

#include "precision.cuh"

__global__ void phiCalc(
    dfloat *phi, dfloat *g, int gpoints,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    #define IDX3D(i,j,k) ((i) + nx * ((j) + ny * (k)))
    #define IDX4D(i,j,k,l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))

    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {
        phi[IDX3D(i,j,k)] = g[IDX4D(i,j,k,0)] + g[IDX4D(i,j,k,1)] + g[IDX4D(i,j,k,2)] +
                            g[IDX4D(i,j,k,3)] + g[IDX4D(i,j,k,4)] + g[IDX4D(i,j,k,5)] +
                            g[IDX4D(i,j,k,6)] + g[IDX4D(i,j,k,7)] + g[IDX4D(i,j,k,8)] +
                            g[IDX4D(i,j,k,9)] + g[IDX4D(i,j,k,10)] + g[IDX4D(i,j,k,11)] +
                            g[IDX4D(i,j,k,12)] + g[IDX4D(i,j,k,13)] + g[IDX4D(i,j,k,14)];
    }

}

__global__ void gradCalc(
    dfloat *phi, dfloat *mod_grad, dfloat *normx, dfloat *normy,
    dfloat *normz, dfloat *indicator, dfloat *w, const dfloat *cix,
    const dfloat *ciy, const dfloat *ciz, int fpoints,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    #define IDX3D(i,j,k) ((i) + nx * ((j) + ny * (k)))

    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {

        dfloat grad_fix = 0, grad_fiy = 0, grad_fiz = 0;
        for (int l = 0; l < fpoints; ++l) {
            grad_fix += 3 * w[l] * cix[l] * phi[IDX3D(i + static_cast<int>(cix[l]),
                                                        j + static_cast<int>(ciy[l]),
                                                        k + static_cast<int>(ciz[l]))];
            grad_fiy += 3 * w[l] * ciy[l] * phi[IDX3D(i + static_cast<int>(cix[l]),
                                                        j + static_cast<int>(ciy[l]),
                                                        k + static_cast<int>(ciz[l]))];
            grad_fiz += 3 * w[l] * ciz[l] * phi[IDX3D(i + static_cast<int>(cix[l]),
                                                        j + static_cast<int>(ciy[l]),
                                                        k + static_cast<int>(ciz[l]))];
        }
        __syncthreads();
        mod_grad[IDX3D(i,j,k)] = sqrt(pow(grad_fix,2) + pow(grad_fiy,2) + pow(grad_fiz,2));
        normx[IDX3D(i,j,k)] = grad_fix / (mod_grad[IDX3D(i,j,k)] + 1e-9);
        normy[IDX3D(i,j,k)] = grad_fiy / (mod_grad[IDX3D(i,j,k)] + 1e-9);
        normz[IDX3D(i,j,k)] = grad_fiz / (mod_grad[IDX3D(i,j,k)] + 1e-9);
        indicator[IDX3D(i,j,k)] = sqrt(pow(grad_fix,2) + pow(grad_fiy,2) + pow(grad_fiz,2));

    }
}

__global__ void curvatureCalc(
    dfloat *curvature, dfloat *indicator, dfloat *w,
    const dfloat *cix, const dfloat *ciy, const dfloat *ciz,
    dfloat *normx, dfloat *normy, dfloat *normz,
    dfloat *ffx, dfloat *ffy, dfloat *ffz, dfloat sigma,
    int fpoints, int nx, int ny, int nz

) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    #define IDX3D(i,j,k) ((i) + nx * ((j) + ny * (k)))

    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {

        curvature[IDX3D(i,j,k)] = 0;
        for (int l = 0; l < fpoints; ++l) {
            curvature[IDX3D(i,j,k)] -= 3 * w[l] *
               (cix[l] * normx[IDX3D(i + static_cast<int>(cix[l]),
                                    j + static_cast<int>(ciy[l]),
                                    k + static_cast<int>(ciz[l]))] +
                ciy[l] * normy[IDX3D(i + static_cast<int>(cix[l]),
                                    j + static_cast<int>(ciy[l]),
                                    k + static_cast<int>(ciz[l]))] +
                ciz[l] * normz[IDX3D(i + static_cast<int>(cix[l]),
                                    j + static_cast<int>(ciy[l]),
                                    k + static_cast<int>(ciz[l]))]
            );
        }
        __syncthreads();
        ffx[IDX3D(i,j,k)] = sigma * curvature[IDX3D(i,j,k)] * normx[IDX3D(i,j,k)] * indicator[IDX3D(i,j,k)];
        ffy[IDX3D(i,j,k)] = sigma * curvature[IDX3D(i,j,k)] * normy[IDX3D(i,j,k)] * indicator[IDX3D(i,j,k)];
        ffz[IDX3D(i,j,k)] = sigma * curvature[IDX3D(i,j,k)] * normz[IDX3D(i,j,k)] * indicator[IDX3D(i,j,k)];

    }

}

__global__ void momentiCalc(
    dfloat *ux, dfloat *uy, dfloat *uz, dfloat *rho,
    dfloat *ffx, dfloat *ffy, dfloat *ffz, dfloat *w, dfloat *f,
    const dfloat *cix, const dfloat *ciy, const dfloat *ciz,
    dfloat *pxx, dfloat *pyy, dfloat *pzz,
    dfloat *pxy, dfloat *pxz, dfloat *pyz,
    dfloat cssq, int nx, int ny, int nz,
    int fpoints, dfloat *fneq
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    #define IDX3D(i,j,k) ((i) + nx * ((j) + ny * (k)))
    #define IDX4D(i,j,k,l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))
    
    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {

        ux[IDX3D(i,j,k)] = (
            f[IDX4D(i,j,k,1)] - f[IDX4D(i,j,k,2)] + f[IDX4D(i,j,k,7)] - f[IDX4D(i,j,k,8)] + f[IDX4D(i,j,k,9)] - 
            f[IDX4D(i,j,k,10)] + f[IDX4D(i,j,k,13)] - f[IDX4D(i,j,k,14)] + f[IDX4D(i,j,k,15)] - f[IDX4D(i,j,k,16)] 
        ) / rho[IDX3D(i,j,k)] + ffx[IDX3D(i,j,k)] * 0.5 / rho[IDX3D(i,j,k)];
        uy[IDX3D(i,j,k)] = (
            f[IDX4D(i,j,k,3)] - f[IDX4D(i,j,k,4)] + f[IDX4D(i,j,k,7)] - f[IDX4D(i,j,k,8)] + f[IDX4D(i,j,k,11)] - 
            f[IDX4D(i,j,k,12)] - f[IDX4D(i,j,k,13)] + f[IDX4D(i,j,k,14)] + f[IDX4D(i,j,k,17)] - f[IDX4D(i,j,k,18)] 
        ) / rho[IDX3D(i,j,k)] + ffy[IDX3D(i,j,k)] * 0.5 / rho[IDX3D(i,j,k)];
        uz[IDX3D(i,j,k)] = (
            f[IDX4D(i,j,k,5)] - f[IDX4D(i,j,k,6)] + f[IDX4D(i,j,k,9)] - f[IDX4D(i,j,k,10)] + f[IDX4D(i,j,k,11)] - 
            f[IDX4D(i,j,k,12)] - f[IDX4D(i,j,k,15)] + f[IDX4D(i,j,k,16)] - f[IDX4D(i,j,k,17)] + f[IDX4D(i,j,k,18)]
        ) / rho[IDX3D(i,j,k)] + ffz[IDX3D(i,j,k)] * 0.5 / rho[IDX3D(i,j,k)];
        __syncthreads();
        dfloat uu = 0.5 * (pow(ux[IDX3D(i,j,k)],2) + pow(uy[IDX3D(i,j,k)],2) + pow(uz[IDX3D(i,j,k)],2)) / cssq;
        rho[IDX3D(i,j,k)] = f[IDX4D(i,j,k,0)] + f[IDX4D(i,j,k,1)] + f[IDX4D(i,j,k,2)] +
                            f[IDX4D(i,j,k,3)] + f[IDX4D(i,j,k,4)] + f[IDX4D(i,j,k,5)] +
                            f[IDX4D(i,j,k,6)] + f[IDX4D(i,j,k,7)] + f[IDX4D(i,j,k,8)] +
                            f[IDX4D(i,j,k,9)] + f[IDX4D(i,j,k,10)] + f[IDX4D(i,j,k,11)] +
                            f[IDX4D(i,j,k,12)] + f[IDX4D(i,j,k,13)] + f[IDX4D(i,j,k,14)] +
                            f[IDX4D(i,j,k,15)] + f[IDX4D(i,j,k,16)] + f[IDX4D(i,j,k,17)] +
                            f[IDX4D(i,j,k,18)]; 
        __syncthreads();
        for (int l = 0; l < fpoints; ++l) {
            dfloat udotc = (ux[IDX3D(i,j,k)] * cix[l] + uy[IDX3D(i,j,k)] * ciy[l] + uz[IDX3D(i,j,k)] * ciz[l]) / cssq;
            dfloat HeF = (w[l] * (rho[IDX3D(i,j,k)] + rho[IDX3D(i,j,k)] * (udotc + 0.5 * pow(udotc,2) - uu)))
                    * ((cix[l] - ux[IDX3D(i,j,k)]) * ffx[IDX3D(i,j,k)] +
                        (ciy[l] - uy[IDX3D(i,j,k)]) * ffy[IDX3D(i,j,k)] +
                        (ciz[l] - uz[IDX3D(i,j,k)]) * ffz[IDX3D(i,j,k)]
                    ) / (rho[IDX3D(i,j,k)] * cssq);
            dfloat feq = w[l] * (rho[IDX3D(i,j,k)] + rho[IDX3D(i,j,k)] * (udotc + 0.5 * pow(udotc,2) - uu)) - 0.5 * HeF;
            fneq[l] = f[IDX4D(i,j,k,l)] - feq;
        }
        __syncthreads();
        pxx[IDX3D(i,j,k)] = fneq[1] + fneq[2] + fneq[7] + fneq[8] + fneq[9] + fneq[10] + fneq[13] + fneq[14] + fneq[15] + fneq[16];
        pyy[IDX3D(i,j,k)] = fneq[3] + fneq[4] + fneq[7] + fneq[8] + fneq[11] + fneq[12] + fneq[13] + fneq[14] + fneq[17] + fneq[18];
        pzz[IDX3D(i,j,k)] = fneq[5] + fneq[6] + fneq[9] + fneq[10] + fneq[11] + fneq[12] + fneq[15] + fneq[16] + fneq[17] + fneq[18];
        pxy[IDX3D(i,j,k)] = fneq[7] + fneq[8] - fneq[13] - fneq[14];
        pxz[IDX3D(i,j,k)] = fneq[9] + fneq[10] - fneq[15] - fneq[16];
        pyz[IDX3D(i,j,k)] = fneq[11] + fneq[12] - fneq[17] - fneq[18];

    }
}

__global__ void collisionCalc(
    dfloat *ux, dfloat *uy, dfloat *uz, dfloat *w, dfloat *w_g,
    const dfloat *cix, const dfloat *ciy, const dfloat *ciz,
    dfloat *normx, dfloat *normy, dfloat *normz,
    dfloat *ffx, dfloat *ffy, dfloat *ffz,
    dfloat *rho, dfloat *phi, dfloat *f, dfloat *g,
    dfloat *pxx, dfloat *pyy, dfloat *pzz, dfloat *pxy, dfloat *pxz, dfloat *pyz,
    dfloat cssq, dfloat omega, dfloat sharp_c, int fpoints, int gpoints,
    int nx, int ny, int nz
) {     
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    #define IDX3D(i,j,k) ((i) + nx * ((j) + ny * (k)))
    #define IDX4D(i,j,k,l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))

    if (i > 0 && i < nx-1 && j > 0 && j < ny-1 && k > 0 && k < nz-1) {

        dfloat uu = 0.5 * (pow(ux[IDX3D(i,j,k)],2) + pow(uy[IDX3D(i,j,k)],2) + pow(uz[IDX3D(i,j,k)],2)) / cssq;
        __syncthreads();
        for (int l = 0; l < fpoints; ++l) {
            dfloat udotc = (ux[IDX3D(i,j,k)] * cix[l] + uy[IDX3D(i,j,k)] * ciy[l] + uz[IDX3D(i,j,k)] * ciz[l]) / cssq;
            dfloat feq = w[l] * (rho[IDX3D(i,j,k)] + rho[IDX3D(i,j,k)] * (udotc + 0.5 * pow(udotc, 2) - uu));
            dfloat HeF = 0.5 * (w[l] * (rho[IDX3D(i,j,k)] + rho[IDX3D(i,j,k)] * (udotc + 0.5 * pow(udotc, 2) - uu)))
                    * ((cix[l] - ux[IDX3D(i,j,k)]) * ffx[IDX3D(i,j,k)] +
                        (ciy[l] - uy[IDX3D(i,j,k)]) * ffy[IDX3D(i,j,k)] +
                        (ciz[l] - uz[IDX3D(i,j,k)]) * ffz[IDX3D(i,j,k)]
                    ) / (rho[IDX3D(i,j,k)] * cssq);
            dfloat singlefneq = (cix[l] * cix[l] - cssq) * pxx[IDX3D(i,j,k)] +
                        (ciy[l] * ciy[l] - cssq) * pyy[IDX3D(i,j,k)] +
                        (ciz[l] * ciz[l] - cssq) * pzz[IDX3D(i,j,k)] +
                        2 * cix[l] * ciy[l] * pxy[IDX3D(i,j,k)] +
                        2 * cix[l] * ciz[l] * pxz[IDX3D(i,j,k)] +
                        2 * ciy[l] * ciz[l] * pyz[IDX3D(i,j,k)];
            f[IDX4D(i + static_cast<int>(cix[l]),
                    j + static_cast<int>(ciy[l]),
                    k + static_cast<int>(ciz[l]),
                    l)] = feq + (1 - omega) * (w[l] / (2 * pow(cssq, 2))) * singlefneq + HeF;
        }
        __syncthreads();
        for (int l = 0; l < gpoints; ++l) {
            dfloat udotc = (ux[IDX3D(i,j,k)] * cix[l] + uy[IDX3D(i,j,k)] * ciy[l] + uz[IDX3D(i,j,k)] * ciz[l]) / cssq;
            dfloat feq = w_g[l] * phi[IDX3D(i,j,k)] * (1 + udotc);
            dfloat Hi = sharp_c * phi[IDX3D(i,j,k)] * (1 - phi[IDX3D(i,j,k)]) *
                (cix[l] * normx[IDX3D(i,j,k)] +
                 ciy[l] * normy[IDX3D(i,j,k)] +
                 ciz[l] * normz[IDX3D(i,j,k)]);
            g[IDX4D(i,j,k,l)] = feq + w_g[l] * Hi;
        }

    }
}

__global__ void streamingCalc(
    dfloat *g, const dfloat *cix, const dfloat *ciy, const dfloat *ciz,
    int nx, int ny, int nz, int gpoints
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    #define IDX4D(i,j,k,l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))

    for (int l = 0; l < gpoints; ++l) {
        g[IDX4D(i,j,k,l)] = g[IDX4D(i + static_cast<int>(cix[l]),
                                    j + static_cast<int>(ciy[l]),
                                    k + static_cast<int>(ciz[l]),
                                    l)];
    }
}

__global__ void boundaryConditions(
    dfloat *f, dfloat *g, dfloat *rho, dfloat *phi, dfloat *w, dfloat *w_g,
    const dfloat *cix, const dfloat *ciy, const dfloat *ciz,
    int fpoints, int gpoints, int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    #define IDX3D(i,j,k) ((i) + nx * ((j) + ny * (k)))
    #define IDX4D(i,j,k,l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))

    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == ny-1) {
        for (int l = 0; l < fpoints; ++l) {
            if (i + static_cast<int>(cix[l]) >= 0 && j + static_cast<int>(ciy[l]) >= 0 && k + static_cast<int>(ciz[l]) >= 0) {
                f[IDX4D(i + static_cast<int>(cix[l]),
                        j + static_cast<int>(ciy[l]),
                        k + static_cast<int>(ciz[l]),
                        l)] = rho[IDX3D(i,j,k)] * w[l];
            }
        }
        for (int l = 0; l < gpoints; ++l) {
            if (i + static_cast<int>(cix[l]) >= 0 && j + static_cast<int>(ciy[l]) >= 0 && k + static_cast<int>(ciz[l]) >= 0) {
                g[IDX4D(i + static_cast<int>(cix[l]),
                        j + static_cast<int>(ciy[l]),
                        k + static_cast<int>(ciz[l]),
                        l)] = phi[IDX3D(i,j,k)] * w_g[l];
            }
        }
    }
    __syncthreads();
    phi[IDX3D(i,j,0)] = phi[IDX3D(i,j,1)];
    phi[IDX3D(i,j,nz-1)] = phi[IDX3D(i,j,nz-2)];
    phi[IDX3D(i,0,k)] = phi[IDX3D(i,1,k)];
    phi[IDX3D(i,ny-1,k)] = phi[IDX3D(i,ny-2,k)];
    phi[IDX3D(0,j,k)] = phi[IDX3D(1,j,k)];
    phi[IDX3D(nx-1,j,k)] = phi[IDX3D(nx-2,j,k)];

}
