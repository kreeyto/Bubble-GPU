#include "kernels.cuh"
#include "utils.cuh"
#include "var.cuh"
#include "errorDef.cuh"
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>
#include <iomanip>

#include "precision.cuh"

int main(int argc, char* argv[]) {
    if (argc < 4) {
        std::cerr << "Erro: Uso: " << argv[0] << " <fluid_model> <phase_model> <id>" << std::endl;
        return 1;
    }

    std::string fluid_model = argv[1];
    std::string phase_model = argv[2];
    std::string id = argv[3];

    std::string base_dir = "/home/breno/Desktop/Bubble-GPU/bin/";
    std::string model_dir = base_dir + fluid_model + "_" + phase_model + "/";
    std::string sim_dir = model_dir + id + "/";

    int ret = system(("mkdir -p " + sim_dir).c_str());
    (void)ret; 

    std::string info_file = sim_dir + id + "_info.txt";

    initializeVars();

    int stamp = 1, nsteps = 50;
    std::vector<dfloat> phi_host(nx * ny * nz, 0.0f);

    generateSimulationInfoFile(info_file, nx, ny, nz, stamp, nsteps, tau, id, fluid_model);

    std::vector<dfloat> f(nx * ny * nz * fpoints, 0.0f);
    std::vector<dfloat> g(nx * ny * nz * gpoints, 0.0f);
    std::vector<dfloat> phi(nx * ny * nz, 0.0f);
    std::vector<dfloat> rho(nx * ny * nz, 1.0f); 
    
    #ifdef FD3Q19
        const std::vector<dfloat> w = {
            1.0f / 3.0f, 
            1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f,
            1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f
        };
    #elif defined(FD3Q27)
        const std::vector <dfloat> w = {
            8.0f / 27.0f,
            2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 
            1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 
            1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f
        };
    #endif

    #ifdef PD3Q15
        const std::vector<dfloat> w_g = {
            2.0f / 9.0f, 
            1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f,
            1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f
        };
    #elif defined(PD3Q19)
        const std::vector <dfloat> w_g = {
            1.0f / 3.0f, 
            1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f,
            1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f
        };
    #elif defined(PD3Q27)
        const std::vector <dfloat> w_g = {
            8.0f / 27.0f,
            2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 
            1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 
            1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f
        };
    #endif

    computeInitialCPU(phi, rho, w, w_g, f, g, nx, ny, nz, fpoints, gpoints, res);
    dfloat max_val = 0.0f;
    for (int k = 0; k < nz; ++k) {
        for (int j = 0; j < ny; ++j) {
            for (int i = 0; i < nx; ++i) {
                dfloat val = phi[i + nx*(j + ny*k)];
                if (val > max_val) max_val = val;
            }
        }
    }
    std::cout << "Valor maximo inicial de phi = " << max_val << std::endl;
    checkCudaErrors(hipMemcpy(d_f, f.data(), nx * ny * nz * fpoints * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_g, g.data(), nx * ny * nz * gpoints * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_phi, phi.data(), nx * ny * nz * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rho, rho.data(), nx * ny * nz * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_w, w.data(), fpoints * sizeof(dfloat), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_w_g, w_g.data(), gpoints * sizeof(dfloat), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(8, 8, 8);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

    for (int t = 0; t < nsteps; ++t) {

        std::cout << "Passo " << t << " de " << nsteps << " iniciado..." << std::endl;

        phiCalc<<<numBlocks, threadsPerBlock>>> (
            d_phi, d_g, nx, ny, nz
        );
        getLastCudaError("Erro no kernel phiCalc");
        hipDeviceSynchronize();

        gradCalc<<<numBlocks, threadsPerBlock>>> (
            d_phi, d_mod_grad, d_normx, d_normy, d_normz, 
            d_indicator, d_w, d_cix, d_ciy, d_ciz, 
            fpoints, nx, ny, nz
        );
        getLastCudaError("Erro no kernel gradCalc");
        hipDeviceSynchronize();

        curvatureCalc<<<numBlocks, threadsPerBlock>>> (
            d_curvature, d_indicator, d_w,
            d_cix, d_ciy, d_ciz,
            d_normx, d_normy, d_normz, 
            d_ffx, d_ffy, d_ffz, sigma,
            fpoints, nx, ny, nz
        );
        getLastCudaError("Erro no kernel curvatureCalc");
        hipDeviceSynchronize();

        momentiCalc<<<numBlocks, threadsPerBlock>>> (
            d_ux, d_uy, d_uz, d_rho,
            d_ffx, d_ffy, d_ffz, d_w, d_f,
            d_cix, d_ciy, d_ciz, 
            d_pxx, d_pyy, d_pzz,
            d_pxy, d_pxz, d_pyz,
            cssq, nx, ny, nz,
            fpoints, d_fneq
        );
        getLastCudaError("Erro no kernel momentiCalc");
        hipDeviceSynchronize();

        collisionCalc<<<numBlocks, threadsPerBlock>>> (
            d_ux, d_uy, d_uz, d_w, d_w_g,
            d_cix, d_ciy, d_ciz,
            d_normx, d_normy, d_normz,
            d_ffx, d_ffy, d_ffz,
            d_rho, d_phi, d_f, d_g, 
            d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, 
            cssq, omega, sharp_c, fpoints, gpoints,
            nx, ny, nz
        );
        getLastCudaError("Erro no kernel collisionCalc");
        hipDeviceSynchronize();

        streamingCalc<<<numBlocks, threadsPerBlock>>> (
            d_g, d_cix, d_ciy, d_ciz, nx, ny, nz, gpoints
        );
        getLastCudaError("Erro no kernel streamingCalc");
        hipDeviceSynchronize();

        boundaryConditions<<<numBlocks, threadsPerBlock>>> (
            d_f, d_g, d_rho, d_phi, d_w, d_w_g,
            d_cix, d_ciy, d_ciz, fpoints, gpoints, nx, ny, nz
        );
        getLastCudaError("Erro no kernel boundaryConditions");
        hipDeviceSynchronize();

        if (t % stamp == 0) {

            std::ostringstream filename_phi;
            checkCudaErrors(hipMemcpy(phi_host.data(), d_phi, nx * ny * nz * sizeof(dfloat), hipMemcpyDeviceToHost));
            filename_phi << sim_dir << id << "_phi" << std::setw(6) << std::setfill('0') << t << ".bin";
            std::ofstream file_phi(filename_phi.str(), std::ios::binary);
            file_phi.write(reinterpret_cast<const char*>(phi_host.data()), phi_host.size() * sizeof(dfloat));
            file_phi.close();

            std::cout << "Passo " << t << ": Dados salvos em " << sim_dir << std::endl;

        }
        
    }

    dfloat *pointers[] = {d_f, d_g, d_phi, d_rho, d_w, d_w_g, d_cix, d_ciy, d_ciz, 
                     d_mod_grad, d_normx, d_normy, d_normz, d_indicator,
                     d_curvature, d_ffx, d_ffy, d_ffz, d_ux, d_uy, d_uz,
                     d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, d_fneq
                     };
    freeMemory(pointers, 28);  

    return 0;
}
