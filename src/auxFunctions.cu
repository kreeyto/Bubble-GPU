#include "hip/hip_runtime.h"
#include "auxFunctions.cuh"
#include "var.cuh"
#include <hip/hip_runtime.h>
#include <fstream>
#include <vector>
#include <iomanip>
#include <string>
#include <cstdlib>
#include <stdexcept>

#include "precision.cuh"

void freeMemory(dfloat **pointers, int count) {
    for (int i = 0; i < count; ++i) {
        if (pointers[i] != nullptr) {
            hipFree(pointers[i]);
        }
    }
}

void computeInitialCPU(
    std::vector<dfloat> &phi, std::vector<dfloat> &rho, const std::vector<dfloat> &w, const std::vector<dfloat> &w_g, 
    std::vector<dfloat> &f, std::vector<dfloat> &g, int nx, int ny, int nz, int fpoints, int gpoints, dfloat res
) {

    auto IDX3D = [&](int i, int j, int k) {
        return ((i) + nx * ((j) + ny * (k)));
    };
    auto IDX4D = [&](int i, int j, int k, int l) {
        return ((i) + nx * ((j) + ny * ((k) + nz * (l))));
    };

    for (int k = 1; k < nz-1; ++k) {
        for (int j = 1; j < ny-1; ++j) {
            for (int i = 1; i < nx-1; ++i) {
                dfloat Ri = std::sqrt((i - nx / 2.0) * (i - nx / 2.0) / 4.0 +
                                        (j - ny / 2.0) * (j - ny / 2.0) +
                                        (k - nz / 2.0) * (k - nz / 2.0));
                phi[IDX3D(i,j,k)] = 0.5 + 0.5 * std::tanh(2.0 * (20 - Ri) / (3.0 * res));
            }
        }
    }

    for (int k = 0; k < nz; ++k) {
        for (int j = 0; j < ny; ++j) {
            for (int i = 0; i < nx; ++i) {
                for (int l = 0; l < fpoints; ++l) {
                    f[IDX4D(i,j,k,l)] = w[l] * rho[IDX3D(i,j,k)];
                }
                for (int l = 0; l < gpoints; ++l) {
                    g[IDX4D(i,j,k,l)] = w_g[l] * phi[IDX3D(i,j,k)];
                }
            }
        }
    }

}

void generateSimulationInfoFile(
    const std::string& filepath, int nx, int ny, int nz, int stamp, int nsteps, dfloat tau, 
    const std::string& sim_id, const std::string& fluid_model
) {
    try {
        std::ofstream file(filepath);

        if (!file.is_open()) {
            std::cerr << "Erro ao abrir o arquivo: " << filepath << std::endl;
            return;
        }

        file << "---------------------------- SIMULATION INFORMATION ----------------------------\n"
             << "                           Simulation ID: " << sim_id << '\n'
             << "                           Velocity set: " << fluid_model << '\n'
             << "                           Precision: " << PRECISION_TYPE << '\n'
             << "                           NX: " << nx << '\n'
             << "                           NY: " << ny << '\n'
             << "                           NZ: " << nz << '\n'
             << "                           NZ_TOTAL: " << nz << '\n'
             << "                           Tau: " << tau << '\n'
             << "                           Umax: 0.000000e+00\n"
             << "                           FX: 0.000000e+00\n"
             << "                           FY: 0.000000e+00\n"
             << "                           FZ: 0.000000e+00\n"
             << "                           Save steps: " << stamp << '\n'
             << "                           Nsteps: " << nsteps << '\n'
             << "                           MLUPS: 1.187970e+01\n"
             << "--------------------------------------------------------------------------------\n";

        file.close();
        std::cout << "Arquivo de informações da simulação criado em: " << filepath << std::endl;
    } catch (const std::exception& e) {
        std::cerr << "Erro ao gerar o arquivo de informações: " << e.what() << std::endl;
    }
}
