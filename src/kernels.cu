#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "var.cuh"
#include <math.h>

#include "precision.h"

#define IDX3D(i,j,k) ((i) + (j) * nx + (k) * nx * ny)
#define IDX4D(i,j,k,l) ((i) + (j) * nx + (k) * nx * ny + (l) * nx * ny * nz)

// ============================================================================================== //

__global__ void initPhase(
    dfloat * __restrict__ phi, 
    int res, int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat bubble_radius = 20.0 * nx / 150.0;

    dfloat dx = i - nx * 0.5;
    dfloat dy = j - ny * 0.5;
    dfloat dz = k - nz * 0.5;
    dfloat Ri = sqrt((dx * dx) / 4.0 + dy * dy + dz * dz);

    dfloat phi_val = 0.5 + 0.5 * tanh(2.0 * (bubble_radius - Ri) / (3.0 * res));

    phi[idx3D] = phi_val;
}


__global__ void initDist(
    const dfloat * __restrict__ rho, 
    const dfloat * __restrict__ phi, 
    dfloat * __restrict__ f,
    dfloat * __restrict__ g,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat rho_val = rho[idx3D];
    dfloat phi_val = phi[idx3D];

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        f[idx4D] = W[l] * rho_val;
    }

    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        g[idx4D] = W_G[l] * phi_val;
    }
}


// ============================================================================================== //

__global__ void phiCalc(
    dfloat * __restrict__ phi,
    const dfloat * __restrict__ g,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat sum = 0.0;       
    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        sum += g[idx4D];
    }

    phi[idx3D] = sum;
}

__global__ void gradCalc(
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ mod_grad,
    dfloat * __restrict__ normx,
    dfloat * __restrict__ normy,
    dfloat * __restrict__ normz,
    dfloat * __restrict__ indicator,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat grad_fix = 0.0, grad_fiy = 0.0, grad_fiz = 0.0;
    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int ii = i + CIX[l];
        int jj = j + CIY[l];
        int kk = k + CIZ[l];
        int offset = ii + jj * nx + kk * nx * ny;
        dfloat val = phi[offset];
        dfloat coef = 3.0 * W[l];
        grad_fix += coef * CIX[l] * val;
        grad_fiy += coef * CIY[l] * val;
        grad_fiz += coef * CIZ[l] * val;
    }

    dfloat gmag_sq = grad_fix * grad_fix + grad_fiy * grad_fiy + grad_fiz * grad_fiz;
    dfloat gmag = sqrt(gmag_sq);

    mod_grad[idx3D] = gmag;
    normx[idx3D] = grad_fix / (gmag + 1e-9);
    normy[idx3D] = grad_fiy / (gmag + 1e-9);
    normz[idx3D] = grad_fiz / (gmag + 1e-9);
    indicator[idx3D] = gmag;
}

__global__ void curvatureCalc(
    dfloat * __restrict__ curvature,
    const dfloat * __restrict__ indicator,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    dfloat * __restrict__ ffx,
    dfloat * __restrict__ ffy,
    dfloat * __restrict__ ffz,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat normx_ = normx[idx3D];
    dfloat normy_ = normy[idx3D];
    dfloat normz_ = normz[idx3D];
    dfloat ind_ = indicator[idx3D];
    dfloat curv = 0.0;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int ii = i + CIX[l];
        int jj = j + CIY[l];
        int kk = k + CIZ[l];
        int offset = ii + jj * nx + kk * nx * ny;
        dfloat normxN = normx[offset];
        dfloat normyN = normy[offset];
        dfloat normzN = normz[offset];
        dfloat coef = 3.0 * W[l];
        curv -= coef * (CIX[l] * normxN + CIY[l] * normyN + CIZ[l] * normzN);
    }

    dfloat mult = SIGMA * curv * ind_;
    curvature[idx3D] = curv;
    ffx[idx3D] = mult * normx_;
    ffy[idx3D] = mult * normy_;
    ffz[idx3D] = mult * normz_;
}

// ============================= blindados de constancia ============================= //

__global__ void momentiCalc(
    dfloat * __restrict__ ux,
    dfloat * __restrict__ uy,
    dfloat * __restrict__ uz,
    dfloat * __restrict__ rho,
    dfloat * __restrict__ ffx,
    dfloat * __restrict__ ffy,
    dfloat * __restrict__ ffz,
    const dfloat * __restrict__ f,
    dfloat * __restrict__ pxx,
    dfloat * __restrict__ pyy,
    dfloat * __restrict__ pzz,
    dfloat * __restrict__ pxy,
    dfloat * __restrict__ pxz,
    dfloat * __restrict__ pyz,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;
    
    dfloat fneq[FPOINTS];
    dfloat fVal[FPOINTS];

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        fVal[l] = __ldg(&f[idx4D]);
    }
    
    dfloat rhoOld = rho[idx3D];
    dfloat ffx_val = ffx[idx3D];
    dfloat ffy_val = ffy[idx3D];
    dfloat ffz_val = ffz[idx3D];

    #ifdef FD3Q19
        dfloat sumUx = fVal[1] - fVal[2] + fVal[7] - fVal[8] +
                        fVal[9] - fVal[10] + fVal[13] - fVal[14] +
                        fVal[15] - fVal[16];

        dfloat sumUy = fVal[3] - fVal[4] + fVal[7] - fVal[8] +
                        fVal[11] - fVal[12] - fVal[13] + fVal[14] +
                        fVal[17] - fVal[18];

        dfloat sumUz = fVal[5] - fVal[6] + fVal[9] - fVal[10] +
                        fVal[11] - fVal[12] - fVal[15] + fVal[16] -
                        fVal[17] + fVal[18];
    #elif defined(FD3Q27)
        dfloat sumUx = fVal[1] - fVal[2] + fVal[7] - fVal[8] + fVal[9] - 
                        fVal[10] + fVal[13] - fVal[14] + fVal[15] - 
                        fVal[16] + fVal[19] - fVal[20] + fVal[21] - 
                        fVal[22] + fVal[23] - fVal[24] - fVal[25] + 
                        fVal[26];

        dfloat sumUy = fVal[3] - fVal[4] + fVal[7] - fVal[8] + fVal[11] - 
                        fVal[12] - fVal[13] + fVal[14] + fVal[17] - 
                        fVal[18] + fVal[19] - fVal[20] + fVal[21] - 
                        fVal[22] - fVal[23] + fVal[24] + fVal[25] - 
                        fVal[26];
                        
        dfloat sumUz = fVal[5] - fVal[6] + fVal[9] - fVal[10] + fVal[11] - 
                        fVal[12] - fVal[15] + fVal[16] - fVal[17] + 
                        fVal[18] + fVal[19] - fVal[20] - fVal[21] + 
                        fVal[22] + fVal[23] - fVal[24] + fVal[25] - 
                        fVal[26];
    #endif

    dfloat invRhoOld = 1.0 / rhoOld;
    dfloat halfFx = 0.5 * ffx_val * invRhoOld;
    dfloat halfFy = 0.5 * ffy_val * invRhoOld;
    dfloat halfFz = 0.5 * ffz_val * invRhoOld;

    dfloat uxVal = sumUx * invRhoOld + halfFx;
    dfloat uyVal = sumUy * invRhoOld + halfFy;
    dfloat uzVal = sumUz * invRhoOld + halfFz;

    dfloat rhoNew = 0.0;
    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l)
        rhoNew += fVal[l];
    rho[idx3D] = rhoNew;

    dfloat invCssq = 1.0 / CSSQ;
    dfloat uu = 0.5 * (uxVal * uxVal + uyVal * uyVal + uzVal * uzVal) * invCssq;
    dfloat invRhoNewCssq = 1.0 / (rhoNew * CSSQ);

    dfloat sumXX = 0.0, sumYY = 0.0, sumZZ = 0.0;
    dfloat sumXY = 0.0, sumXZ = 0.0, sumYZ = 0.0;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        dfloat udotc = (uxVal * CIX[l] + uyVal * CIY[l] + uzVal * CIZ[l]) * invCssq;
        dfloat udotc2 = udotc * udotc;
        dfloat eqBase = rhoNew * (udotc + 0.5 * udotc2 - uu);
        dfloat common = W[l] * (rhoNew + eqBase);
        dfloat feq = common;
        dfloat HeF = common * ((CIX[l] - uxVal) * ffx_val +
                               (CIY[l] - uyVal) * ffy_val +
                               (CIZ[l] - uzVal) * ffz_val) * invRhoNewCssq;
        feq -= 0.5 * HeF;
        fneq[l] = fVal[l] - feq;
    }

    pxx[idx3D] = sumXX;
    pyy[idx3D] = sumYY;
    pzz[idx3D] = sumZZ;
    pxy[idx3D] = sumXY;
    pxz[idx3D] = sumXZ;
    pyz[idx3D] = sumYZ;

    ux[idx3D] = uxVal;
    uy[idx3D] = uyVal;
    uz[idx3D] = uzVal;
}

__global__ void collisionCalc(
    const dfloat * __restrict__ ux,
    const dfloat * __restrict__ uy,
    const dfloat * __restrict__ uz,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    const dfloat * __restrict__ ffx,
    const dfloat * __restrict__ ffy,
    const dfloat * __restrict__ ffz,
    const dfloat * __restrict__ rho,
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ g,
    const dfloat * __restrict__ pxx,
    const dfloat * __restrict__ pyy,
    const dfloat * __restrict__ pzz,
    const dfloat * __restrict__ pxy,
    const dfloat * __restrict__ pxz,
    const dfloat * __restrict__ pyz,
    int nx, int ny, int nz,
    dfloat * __restrict__ f_coll
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1 || k == 0 || k == nz - 1) return;

    int idx3D = i + j * nx + k * nx * ny;
    int nxyz = nx * ny * nz;

    dfloat ux_val = ux[idx3D];
    dfloat uy_val = uy[idx3D];
    dfloat uz_val = uz[idx3D];
    dfloat rho_val = rho[idx3D];
    dfloat phi_val = phi[idx3D];
    dfloat ffx_val = ffx[idx3D];
    dfloat ffy_val = ffy[idx3D];
    dfloat ffz_val = ffz[idx3D];
    dfloat pxx_val = pxx[idx3D];
    dfloat pyy_val = pyy[idx3D]; 
    dfloat pzz_val = pzz[idx3D];
    dfloat pxy_val = pxy[idx3D];
    dfloat pxz_val = pxz[idx3D];
    dfloat pyz_val = pyz[idx3D];
    dfloat normx_val = normx[idx3D];
    dfloat normy_val = normy[idx3D];
    dfloat normz_val = normz[idx3D];

    dfloat uu = 0.5 * (ux_val * ux_val + uy_val * uy_val + uz_val * uz_val) / CSSQ;
    dfloat one_minus_omega = 1.0 - OMEGA;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) / CSSQ;
        dfloat feq = W[l] * (rho_val + rho_val * (udotc + 0.5 * udotc * udotc - uu));
        dfloat HeF = 0.5 * feq *
                    ((CIX[l] - ux_val) * ffx_val +
                     (CIY[l] - uy_val) * ffy_val +
                     (CIZ[l] - uz_val) * ffz_val) / (rho_val * CSSQ);
        dfloat fneq = (CIX[l] * CIX[l] - CSSQ) * pxx_val +
                      (CIY[l] * CIY[l] - CSSQ) * pyy_val +
                      (CIZ[l] * CIZ[l] - CSSQ) * pzz_val +
                       2 * CIX[l] * CIY[l] * pxy_val +
                       2 * CIX[l] * CIZ[l] * pxz_val +
                       2 * CIY[l] * CIZ[l] * pyz_val;
        f_coll[idx3D + l * nxyz] = feq + one_minus_omega * (W[l] / (2.0 * CSSQ * CSSQ)) * fneq + HeF;
    }
    
    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) / CSSQ;
        dfloat feq = W_G[l] * phi_val * (1 + udotc);
        dfloat Hi = SHARP_C * phi_val * (1 - phi_val) *
                        (CIX[l] * normx_val + CIY[l] * normy_val + CIZ[l] * normz_val);
        g[idx3D + l * nxyz] = feq + W_G[l] * Hi;
    }
}

// ============================= fim blindados de constancia ============================= //

__global__ void streamingCalcNew(
    const dfloat * __restrict__ f_coll,
    int nx, int ny, int nz,
    dfloat * __restrict__ f 
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int NxNy = nx * ny;
    int NxNyNz = NxNy * nz;
    int dstBase = i + j * nx + k * NxNy;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int src_i = (i - CIX[l] + nx) & (nx-1);
        int src_j = (j - CIY[l] + ny) & (ny-1);
        int src_k = (k - CIZ[l] + nz) & (nz-1);
        int srcBase = src_i + src_j * nx + src_k * NxNy;
        int dstIdx = l * NxNyNz + dstBase;
        int srcIdx = l * NxNyNz + srcBase;
        f[dstIdx] = f_coll[srcIdx];
    }
}

__global__ void streamingCalc(
    const dfloat * __restrict__ g_in,
    dfloat * __restrict__ g_out,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int NxNy = nx * ny;
    int NxNyNz = NxNy * nz;
    int dstBase = i + j * nx + k * NxNy;

    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        int src_i = (i - CIX[l] + nx) & (nx-1);
        int src_j = (j - CIY[l] + ny) & (ny-1);
        int src_k = (k - CIZ[l] + nz) & (nz-1);
        int srcBase = src_i + src_j * nx + src_k * NxNy;
        int dstIdx = l * NxNyNz + dstBase;
        int srcIdx = l * NxNyNz + srcBase;
        g_out[dstIdx] = g_in[srcIdx];
    }
}

/*
__global__ void fgBoundary(
    dfloat * __restrict__ f,
    dfloat * __restrict__ g,
    const dfloat * __restrict__ rho,
    const dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z; 

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = i + j * nx + k * nx * ny;

    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) {
        #pragma unroll 19
        for (int l = 0; l < FPOINTS; ++l) {
            int nb_i = i + CIX[l];
            int nb_j = j + CIY[l];
            int nb_k = k + CIZ[l];
            if (nb_i >= 0 && nb_i < nx && nb_j >= 0 && nb_j < ny && nb_k >= 0 && nb_k < nz) {
                f[IDX4D(nb_i,nb_j,nb_k,l)] = rho[idx3D] * W[l];
            }
        }
        #pragma unroll 15
        for (int l = 0; l < GPOINTS; ++l) {
            int nb_i = i + CIX[l];
            int nb_j = j + CIY[l];
            int nb_k = k + CIZ[l];
            if (nb_i >= 0 && nb_i < nx && nb_j >= 0 && nb_j < ny && nb_k >= 0 && nb_k < nz) {
                g[IDX4D(nb_i,nb_j,nb_k,l)] = phi[idx3D] * W_G[l];
            }
        }
    }
}
*/

__global__ void fgBoundary_f(
    dfloat * __restrict__ f,
    const dfloat * __restrict__ rho,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if(i >= nx || j >= ny || k >= nz) return;
    
    for (int l = 0; l < FPOINTS; ++l) {
        int bi = i - CIX[l];
        int bj = j - CIY[l];
        int bk = k - CIZ[l];
        if(bi < 0 || bi >= nx || bj < 0 || bj >= ny || bk < 0 || bk >= nz)
            continue;
        if(bi == 0 || bi == nx-1 || bj == 0 || bj == ny-1 || bk == 0 || bk == nz-1) {
            int boundary_idx = bi + bj * nx + bk * nx * ny;
            f[IDX4D(i, j, k, l)] = rho[boundary_idx] * W[l];
        }
    }
}

__global__ void fgBoundary_g(
    dfloat * __restrict__ g,
    const dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    if(i >= nx || j >= ny || k >= nz) return;
    
    for (int l = 0; l < GPOINTS; ++l) {
        int bi = i - CIX[l];
        int bj = j - CIY[l];
        int bk = k - CIZ[l];
        if(bi < 0 || bi >= nx || bj < 0 || bj >= ny || bk < 0 || bk >= nz)
            continue;
        if(bi == 0 || bi == nx-1 || bj == 0 || bj == ny-1 || bk == 0 || bk == nz-1) {
            int boundary_idx = bi + bj * nx + bk * nx * ny;
            g[IDX4D(i, j, k, l)] = phi[boundary_idx] * W_G[l];
        }
    }
}


__global__ void boundaryConditions_z(
    dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        phi[IDX3D(i,j,0)] = phi[IDX3D(i,j,1)];
        phi[IDX3D(i,j,nz-1)] = phi[IDX3D(i,j,nz-2)];
    }
}

__global__ void boundaryConditions_y(
    dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && k < nz) {
        phi[IDX3D(i,0,k)] = phi[IDX3D(i,1,k)];
        phi[IDX3D(i,ny-1,k)] = phi[IDX3D(i,ny-2,k)];
    }
}