#include "kernels.cuh"
#include "auxFunctions.cuh"
#include "var.cuh"
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>
#include <iomanip>
#include <chrono>

#include "precision.cuh"

int main(int argc, char* argv[]) {
    auto start_time = chrono::high_resolution_clock::now();
    if (argc < 4) {
        cerr << "Erro: Uso: " << argv[0] << " F<fluid velocity set> P<phase field velocity set> <id>" << endl;
        return 1;
    }
    string fluid_model = argv[1];
    string phase_model = argv[2];
    string id = argv[3];

    string base_dir;   
    #ifdef _WIN32
        base_dir = "..\\";
    #else
        base_dir = "../";
    #endif
    string model_dir = base_dir + "bin/" + fluid_model + "_" + phase_model + "/";
    string sim_dir = model_dir + id + "/";
    #ifdef _WIN32
        string mkdir_command = "mkdir \"" + sim_dir + "\"";
    #else
        string mkdir_command = "mkdir -p \"" + sim_dir + "\"";
    #endif
    int ret = system(mkdir_command.c_str());
    (void)ret;

    // ============================================================================================================================================================= //

    // ========================= //
    int stamp = 100, nsteps = 5000;
    // ========================= //
    initializeVars();

    string info_file = sim_dir + id + "_info.txt";
    dfloat h_tau;
    hipMemcpyFromSymbol(&h_tau, HIP_SYMBOL(TAU), sizeof(dfloat), 0, hipMemcpyDeviceToHost);
    generateSimulationInfoFile(info_file, nx, ny, nz, stamp, nsteps, h_tau, id, fluid_model);

    dim3 threadsPerBlock(8,8,8);
    dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

    // STREAMS
    hipStream_t mainStream;
    hipStreamCreate(&mainStream);

    // ================== INIT ================== //

        initTensor<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
            d_pxx, d_pyy, d_pzz, 
            d_pxy, d_pxz, d_pyz,
            d_rho, nx, ny, nz
        );

        initPhase<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
            d_phi, nx, ny, nz
        ); 

        initDist<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
            d_rho, d_phi, d_f, d_g, nx, ny, nz
        ); 

    // ========================================= //

    vector<dfloat> phi_host(nx * ny * nz);
    vector<dfloat> ux_host(nx * ny * nz);
    vector<dfloat> uy_host(nx * ny * nz);
    vector<dfloat> uz_host(nx * ny * nz);

    for (int t = 0; t <= nsteps ; ++t) {
        cout << "Passo " << t << " de " << nsteps << " iniciado..." << endl;



        // ================= PHASE FIELD ================= //

            phiCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_phi, d_g, nx, ny, nz
            ); 

        // =============================================== // 
        


        // ===================== NORMALS ===================== //

            gradCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_phi, d_normx, d_normy, d_normz, 
                d_indicator, 
                nx, ny, nz
            ); 

        // =================================================== // 

        

        // ==================== CURVATURE ==================== //

            curvatureCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_curvature, d_indicator,
                d_normx, d_normy, d_normz, 
                d_ffx, d_ffy, d_ffz,
                nx, ny, nz
            ); 

        // =================================================== //   


        
        // ===================== MOMENTI ===================== //

            momentiCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_ux, d_uy, d_uz, d_rho,
                d_ffx, d_ffy, d_ffz, d_f,
                d_pxx, d_pyy, d_pzz,
                d_pxy, d_pxz, d_pyz,
                nx, ny, nz
            ); 

        // ================================================== //   

        

        // ==================== COLLISION & STREAMING ==================== //
            
            collisionCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_ux, d_uy, d_uz, 
                d_normx, d_normy, d_normz,
                d_ffx, d_ffy, d_ffz,
                d_rho, d_phi, d_g, 
                d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, 
                nx, ny, nz, d_f
            ); 

        // =============================================================== //    



        // =================== STREAMING =================== //
            
            streamingCalc<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_g, d_g_out, 
                nx, ny, nz
            ); 
            hipMemcpy(d_g, d_g_out, nx * ny * nz * GPOINTS * sizeof(dfloat), hipMemcpyDeviceToDevice);

        // ================================================= //



        // ========================================== DISTRIBUTION ========================================== //

            fgBoundary<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_f, d_rho,
                d_g, d_phi,
                nx, ny, nz
            ); 

        // ================================================================================================= //



        // ======================= BOUNDARY ======================= //

            boundaryConditions<<<numBlocks, threadsPerBlock, 0, mainStream>>> (
                d_phi, nx, ny, nz
            ); 

        // ======================================================== //

        hipDeviceSynchronize();

        if (t % stamp == 0) {

            copyAndSaveToBinary(d_phi, nx * ny * nz, sim_dir, id, t, "phi");
            copyAndSaveToBinary(d_ux, nx * ny * nz, sim_dir, id, t, "ux");
            copyAndSaveToBinary(d_uy, nx * ny * nz, sim_dir, id, t, "uy");
            copyAndSaveToBinary(d_uz, nx * ny * nz, sim_dir, id, t, "uz");

            cout << "Passo " << t << ": Dados salvos em " << sim_dir << endl;
        }
    }

    hipStreamDestroy(mainStream);

    dfloat *pointers[] = {d_f, d_g, d_phi, d_rho, 
                          d_normx, d_normy, d_normz, d_indicator,
                          d_curvature, d_ffx, d_ffy, d_ffz, d_ux, d_uy, d_uz,
                          d_pxx, d_pyy, d_pzz, d_pxy, d_pxz, d_pyz, d_g_out
                        };
    freeMemory(pointers, 22);  

    auto end_time = chrono::high_resolution_clock::now();
    chrono::duration<double> elapsed_time = end_time - start_time;
    cout << "Tempo total de execução: " << elapsed_time.count() << " segundos" << endl;

    return 0;
}
