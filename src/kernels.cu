#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "constants.cuh"
#include <math.h>

// FIRST PART

__global__ void momentCalc(
    const float *f, const float *rho, const float *ffx, const float *ffy, const float *ffz,
    const float *cix, const float *ciy, const float *ciz,
    float *ux, float *uy, float *uz, float *pxx, float *pyy, float *pzz,
    float *pxy, float *pxz, float *pyz,
    int nx, int ny, int nz, int fpoints, float cssq, const float *w) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    int idx = i + nx * (j + ny * k);
    #define F_IDX(i, j, k, l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))

    if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1 && k > 0 && k < nz - 1) {
        ux[idx] = (
            (f[F_IDX(i, j, k, 1)] + f[F_IDX(i, j, k, 15)] + f[F_IDX(i, j, k, 9)] + f[F_IDX(i, j, k, 7)] + f[F_IDX(i, j, k, 13)]) -
            (f[F_IDX(i, j, k, 2)] + f[F_IDX(i, j, k, 10)] + f[F_IDX(i, j, k, 16)] + f[F_IDX(i, j, k, 14)] + f[F_IDX(i, j, k, 7)])
        ) / rho[idx] +
        ffx[idx] * 0.5 / rho[idx];

        uy[idx] = (
            (f[F_IDX(i, j, k, 3)] + f[F_IDX(i, j, k, 7)] + f[F_IDX(i, j, k, 14)] + f[F_IDX(i, j, k, 17)] + f[F_IDX(i, j, k, 11)]) -
            (f[F_IDX(i, j, k, 4)] + f[F_IDX(i, j, k, 13)] + f[F_IDX(i, j, k, 8)] + f[F_IDX(i, j, k, 12)] + f[F_IDX(i, j, k, 18)])
        ) / rho[idx] +
        ffy[idx] * 0.5 / rho[idx];

        uz[idx] = (
            (f[F_IDX(i, j, k, 6)] + f[F_IDX(i, j, k, 15)] + f[F_IDX(i, j, k, 10)] + f[F_IDX(i, j, k, 17)] + f[F_IDX(i, j, k, 12)]) -
            (f[F_IDX(i, j, k, 5)] + f[F_IDX(i, j, k, 9)] + f[F_IDX(i, j, k, 16)] + f[F_IDX(i, j, k, 11)] + f[F_IDX(i, j, k, 18)])
        ) / rho[idx] +
        ffz[idx] * 0.5 / rho[idx];

        double fneq[19];
        double uu = 0.5 * (pow(ux[idx], 2) + pow(uy[idx], 2) + pow(uz[idx], 2)) / cssq;

        for (int l = 0; l < fpoints; l++) {
            double udotc = (ux[idx] * cix[l] + uy[idx] * ciy[l] + uz[idx] * ciz[l]) / cssq;
            double HeF = (w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc, 2) - uu)))
                     * ((cix[l] - ux[idx]) * ffx[idx] + 
                        (ciy[l] - uy[idx]) * ffy[idx] + 
                        (ciz[l] - uz[idx]) * ffz[idx] 
                       ) / (rho[idx] * cssq);
            double feq = w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc, 2) - uu)) - 0.5 * HeF;
            fneq[l] = f[F_IDX(i, j, k, l)] - feq;
        }

        pxx[idx] = fneq[2] + fneq[3] + fneq[8] + fneq[9] + fneq[10] + fneq[11] + fneq[14] + fneq[15] + fneq[16] + fneq[17];
        pyy[idx] = fneq[4] + fneq[5] + fneq[8] + fneq[9] + fneq[12] + fneq[13] + fneq[14] + fneq[15] + fneq[18] + fneq[19];
        pzz[idx] = fneq[6] + fneq[7] + fneq[10] + fneq[11] + fneq[12] + fneq[13] + fneq[16] + fneq[17] + fneq[18] + fneq[19];
        pxy[idx] = fneq[8] + fneq[9] - fneq[14] - fneq[15];
        pxz[idx] = fneq[10] + fneq[11] - fneq[16] - fneq[17];
        pyz[idx] = fneq[12] + fneq[13] - fneq[18] - fneq[19];
    }
}

__global__ void collisionCalc(
    float *f, float *g, const float *phi, const float *rho, const float *ux, const float *uy, const float *uz,
    const float *ffx, const float *ffy, const float *ffz,
    const float *pxx, const float *pyy, const float *pzz, const float *pxy, const float *pxz, const float *pyz,
    const float *cix, const float *ciy, const float *ciz, const float *w, const float *w_g,
    const float *normx, const float *normy, const float *normz,
    int nx, int ny, int nz, int fpoints, int gpoints, float cssq, float omega, float sharp_c) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    int idx = i + nx * (j + ny * k);
    #define F_IDX(i, j, k, l) ((i) + nx * ((j) + ny * ((k) + nz * (l))))

    if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1 && k > 0 && k < nz - 1) {

        double uu = 0.5 * (pow(ux[idx], 2) + pow(uy[idx], 2) + pow(uz[idx], 2)) / cssq;

        for (int l = 0; l < fpoints; l++) {
            double udotc = (ux[idx] * cix[l] + uy[idx] * ciy[l] + uz[idx] * ciz[l]) / cssq;
            double feq = w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc, 2) - uu));
            double HeF = 0.5 * (w[l] * (rho[idx] + rho[idx] * (udotc + 0.5 * pow(udotc, 2) - uu)))
                     * ((cix[l] - ux[idx]) * ffx[idx] + 
                        (ciy[l] - uy[idx]) * ffy[idx] + 
                        (ciz[l] - uz[idx]) * ffz[idx] 
                       ) / (rho[idx] * cssq);
            double fneq = (cix[l] * cix[l] - cssq) * pxx[idx] + 
                          (ciy[l] * ciy[l] - cssq) * pyy[idx] + 
                          (ciz[l] * ciz[l] - cssq) * pzz[idx] + 
                          2 * cix[l] * ciy[l] * pxy[idx] + 
                          2 * cix[l] * ciz[l] * pxz[idx] + 
                          2 * ciy[l] * ciz[l] * pyz[idx];
            f[F_IDX(i + static_cast<int>(cix[l]),
                    j + static_cast<int>(ciy[l]),
                    k + static_cast<int>(ciz[l]),
                    l)] = feq + (1 - omega) * (w[l] / (2 * pow(cssq, 2))) * fneq + HeF;
        }

        for (int l = 0; l < gpoints; l++) {
            double udotc = (ux[idx] * cix[l] + uy[idx] * ciy[l] + uz[idx] * ciz[l]) / cssq;
            double feq = w_g[l] * phi[idx] * (1 + udotc);
            double Hi = sharp_c * phi[idx] * (1 - phi[idx]) * (cix[l] * normx[idx] + ciy[l] * normy[idx] + ciz[l] * normz[idx]); 
            g[F_IDX(i, j, k, l)] = feq + w_g[l] * Hi;
        }

    }
}

// REMAINING PORTION. ADJUST
/*
for l = 1:gpoints
    g(:,:,:,l) = circshift(g(:,:,:,l),[cix(l),ciy(l),ciz(l)]);
end

% boundary conditions
for i = [1,nx]
    for j = [1,ny]
        for k = [1,nz]
            for l = 1:fpoints
                if (i+cix(l)>0 && j+ciy(l)>0 && k+ciz(l)>0)
                    f(i+cix(l),j+ciy(l),k+ciz(l),l) = rho(i,j,k) .* w(l); 
                end
            end
            for l = 1:gpoints
                if (i+cix(l)>0 && j+ciy(l)>0 && k+ciz(l)>0)
                    g(i+cix(l),j+ciy(l),k+ciz(l),l) = phi(i,j,k) .* w_g(l);
                end
            end
        end
    end
end

phi(:,:,1) = phi(:,:,2);  
phi(:,:,nz) = phi(:,:,nz-1); 
phi(1,:,:) = phi(2,:,:); 
phi(nx,:,:) = phi(nx-1,:,:); 
phi(:,1,:) = phi(:,2,:); 
phi(:,ny,:) = phi(:,ny-1,:); 
*/