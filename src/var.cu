#include "var.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include "precision.cuh"

int mesh = 128;
int nx = mesh; int ny = mesh; int nz = mesh;  

__constant__ dfloat TAU;
__constant__ dfloat CSSQ;
__constant__ dfloat OMEGA;
__constant__ dfloat SHARP_C;
__constant__ dfloat SIGMA;
__constant__ dfloat W[FPOINTS], W_G[GPOINTS];
__constant__ int CIX[FPOINTS], CIY[FPOINTS], CIZ[FPOINTS];

dfloat *d_f, *d_g;
dfloat *d_normx, *d_normy, *d_normz, *d_indicator;
dfloat *d_curvature, *d_ffx, *d_ffy, *d_ffz;
dfloat *d_ux, *d_uy, *d_uz, *d_pxx, *d_pyy, *d_pzz;
dfloat *d_pxy, *d_pxz, *d_pyz, *d_rho, *d_phi;
dfloat *d_g_out; // *d_f_coll

// ========================================================================== parametros ========================================================================== //
dfloat h_tau = 0.505;
dfloat h_cssq = 1.0 / 3.0;
dfloat h_omega = 1.0 / h_tau;
dfloat h_sharp_c = 0.15 * 3.0;
dfloat h_sigma = 0.1;

// fluid velocity set
#ifdef FD3Q19
    int h_cix[19] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0 };
    int h_ciy[19] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 0, 0, 1, -1, -1, 1, 0, 0, 1, -1 };
    int h_ciz[19] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, -1, 1, -1, 1 };
#elif defined(FD3Q27)
    int h_cix[27] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 1, -1, -1, 1 };
    int h_ciy[27] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 0, 0, 1, -1, -1, 1, 0, 0, 1, -1, 1, -1, 1, -1, -1, 1, 1, -1 };
    int h_ciz[27] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, -1, 1, -1, 1, 1, -1, -1, 1, 1, -1, 1, -1 };
#endif

// fluid weights
#ifdef FD3Q19
    dfloat h_w[19] = {
        1.0 / 3.0, 
        1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0,
        1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0
    };
#elif defined(FD3Q27)
    dfloat h_w[27] = {
        8.0 / 27.0,
        2.0 / 27.0, 2.0 / 27.0, 2.0 / 27.0, 2.0 / 27.0, 2.0 / 27.0, 2.0 / 27.0, 
        1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 1.0 / 54.0, 
        1.0 / 216.0, 1.0 / 216.0, 1.0 / 216.0, 1.0 / 216.0, 1.0 / 216.0, 1.0 / 216.0, 1.0 / 216.0, 1.0 / 216.0
    };
#endif

// phase field weights
#ifdef PD3Q19
    dfloat h_w_g[19] = {
        1.0 / 3.0, 
        1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0, 1.0 / 18.0,
        1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0
    };
#endif
// =============================================================================================================================================================== //

void initializeVars() {
    size_t size = nx * ny * nz * sizeof(dfloat);            
    size_t f_size = nx * ny * nz * FPOINTS * sizeof(dfloat); 
    size_t g_size = nx * ny * nz * GPOINTS * sizeof(dfloat); 

    hipMalloc((void **)&d_rho, size);
    hipMalloc((void **)&d_phi, size);
    hipMalloc((void **)&d_ux, size);
    hipMalloc((void **)&d_uy, size);
    hipMalloc((void **)&d_uz, size);
    hipMalloc((void **)&d_normx, size);
    hipMalloc((void **)&d_normy, size);
    hipMalloc((void **)&d_normz, size);
    hipMalloc((void **)&d_curvature, size);
    hipMalloc((void **)&d_indicator, size);
    hipMalloc((void **)&d_ffx, size);
    hipMalloc((void **)&d_ffy, size);
    hipMalloc((void **)&d_ffz, size);
    hipMalloc((void **)&d_pxx, size);
    hipMalloc((void **)&d_pyy, size);
    hipMalloc((void **)&d_pzz, size);
    hipMalloc((void **)&d_pxy, size);
    hipMalloc((void **)&d_pxz, size);
    hipMalloc((void **)&d_pyz, size);

    hipMalloc((void **)&d_f, f_size);
    hipMalloc((void **)&d_g, g_size);

    hipMalloc((void **)&d_g_out, g_size);

    hipMemset(d_phi, 0, size);
    hipMemset(d_ux, 0, size);
    hipMemset(d_uy, 0, size);
    hipMemset(d_uz, 0, size);
    
    hipMemset(d_f, 0, f_size);
    hipMemset(d_g, 0, g_size);

    hipMemset(d_normx, 0, size);
    hipMemset(d_normy, 0, size);
    hipMemset(d_normz, 0, size);
    hipMemset(d_curvature, 0, size);
    hipMemset(d_indicator, 0, size);
    hipMemset(d_ffx, 0, size);
    hipMemset(d_ffy, 0, size);
    hipMemset(d_ffz, 0, size);

    hipMemcpyToSymbol(HIP_SYMBOL(TAU), &h_tau, sizeof(dfloat));
    hipMemcpyToSymbol(HIP_SYMBOL(CSSQ), &h_cssq, sizeof(dfloat));
    hipMemcpyToSymbol(HIP_SYMBOL(OMEGA), &h_omega, sizeof(dfloat));
    hipMemcpyToSymbol(HIP_SYMBOL(SHARP_C), &h_sharp_c, sizeof(dfloat));
    hipMemcpyToSymbol(HIP_SYMBOL(SIGMA), &h_sigma, sizeof(dfloat));

    hipMemcpyToSymbol(HIP_SYMBOL(W), &h_w, FPOINTS * sizeof(dfloat));
    hipMemcpyToSymbol(HIP_SYMBOL(W_G), &h_w_g, GPOINTS * sizeof(dfloat));

    hipMemcpyToSymbol(HIP_SYMBOL(CIX), &h_cix, FPOINTS * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(CIY), &h_ciy, FPOINTS * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(CIZ), &h_ciz, FPOINTS * sizeof(int));

}

