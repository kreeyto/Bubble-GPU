#include "constants.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

int nx = 128, ny = 128, nz = 128, fpoints = 19, gpoints = 15, nsteps = 1000;
float tau = 0.505, cssq = 1.0f / 3.0f, omega = 1.0f / tau, sharp_c = 0.45f, sigma = 0.1f;

float *d_f, *d_g, *d_w, *d_w_g, *d_cix, *d_ciy, *d_ciz;
float *d_grad_fix, *d_grad_fiy, *d_grad_fiz, *d_mod_grad;
float *d_normx, *d_normy, *d_normz, *d_indicator;
float *d_curvature, *d_ffx, *d_ffy, *d_ffz;
float *d_ux, *d_uy, *d_uz, *d_pxx, *d_pyy, *d_pzz;
float *d_pxy, *d_pxz, *d_pyz, *d_rho, *d_phi;

float *h_rho = (float *)malloc(nx * ny * nz * sizeof(float));
float *h_pxx = (float *)malloc(nx * ny * nz * sizeof(float));
float *h_pyy = (float *)malloc(nx * ny * nz * sizeof(float));
float *h_pzz = (float *)malloc(nx * ny * nz * sizeof(float));
float *h_pxy = (float *)malloc(nx * ny * nz * sizeof(float));
float *h_pxz = (float *)malloc(nx * ny * nz * sizeof(float));
float *h_pyz = (float *)malloc(nx * ny * nz * sizeof(float));

const float w[19] = {
    1.0f / 3.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f,
    1.0f / 18.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f,
    1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f
};

const float w_g[15] = {
    2.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f,
    1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f,
    1.0f / 72.0f, 1.0f / 72.0f, 1.0f / 72.0f
};

const float cix[19] = { 0, 1, -1, 0, 0, 0,  0,  1, -1,  1, -1,  0,  0,  0,  1, -1,  1, -1,  0 };
const float ciy[19] = { 0, 0,  0, 1, -1, 0,  0,  1,  1, -1, -1,  1, -1,  0,  0,  0,  0,  0, -1 };
const float ciz[19] = { 0, 0,  0, 0,  0, 1, -1,  0,  0,  0,  0,  1,  1, -1, -1,  1,  1, -1, -1 };

void initializeConstants() {
    size_t size = nx * ny * nz * sizeof(float);            
    size_t f_size = nx * ny * nz * fpoints * sizeof(float); 
    size_t g_size = nx * ny * nz * gpoints * sizeof(float); 

    for (int i = 0; i < nx * ny * nz; i++) {
        h_pxx[i] = 1.0f;
        h_pyy[i] = 1.0f;
        h_pzz[i] = 1.0f;
        h_pxy[i] = 1.0f;
        h_pxz[i] = 1.0f;
        h_pyz[i] = 1.0f;
    }   

    hipMalloc((void **)&d_rho, size);
    hipMalloc((void **)&d_phi, size);
    hipMalloc((void **)&d_ux, size);
    hipMalloc((void **)&d_uy, size);
    hipMalloc((void **)&d_uz, size);
    hipMalloc((void **)&d_normx, size);
    hipMalloc((void **)&d_normy, size);
    hipMalloc((void **)&d_normz, size);
    hipMalloc((void **)&d_curvature, size);
    hipMalloc((void **)&d_indicator, size);
    hipMalloc((void **)&d_ffx, size);
    hipMalloc((void **)&d_ffy, size);
    hipMalloc((void **)&d_ffz, size);
    hipMalloc((void **)&d_mod_grad, size);
    hipMalloc((void **)&d_pxx, size);
    hipMalloc((void **)&d_pyy, size);
    hipMalloc((void **)&d_pzz, size);
    hipMalloc((void **)&d_pxy, size);
    hipMalloc((void **)&d_pxz, size);
    hipMalloc((void **)&d_pyz, size);

    hipMalloc((void **)&d_f, f_size);
    hipMalloc((void **)&d_g, g_size);
    hipMalloc((void **)&d_w, fpoints * sizeof(float));
    hipMalloc((void **)&d_w_g, gpoints * sizeof(float));
    hipMalloc((void **)&d_cix, fpoints * sizeof(float));
    hipMalloc((void **)&d_ciy, fpoints * sizeof(float));
    hipMalloc((void **)&d_ciz, fpoints * sizeof(float));

    hipMemset(d_ux, 0, size);
    hipMemset(d_uy, 0, size);
    hipMemset(d_uz, 0, size);
    hipMemset(d_normx, 0, size);
    hipMemset(d_normy, 0, size);
    hipMemset(d_normz, 0, size);
    hipMemset(d_curvature, 0, size);
    hipMemset(d_indicator, 0, size);
    hipMemset(d_ffx, 0, size);
    hipMemset(d_ffy, 0, size);
    hipMemset(d_ffz, 0, size);
    hipMemset(d_mod_grad, 0, size);
    hipMemset(d_f, 0, f_size);
    hipMemset(d_g, 0, g_size);

    hipMemcpy(d_pxx, h_pxx, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pyy, h_pyy, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pzz, h_pzz, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pxy, h_pxy, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pxz, h_pxz, size, hipMemcpyHostToDevice);
    hipMemcpy(d_pyz, h_pyz, size, hipMemcpyHostToDevice);
    hipMemcpy(d_w, w, sizeof(w), hipMemcpyHostToDevice);
    hipMemcpy(d_w_g, w_g, sizeof(w_g), hipMemcpyHostToDevice);
    hipMemcpy(d_cix, cix, sizeof(cix), hipMemcpyHostToDevice);
    hipMemcpy(d_ciy, ciy, sizeof(ciy), hipMemcpyHostToDevice);
    hipMemcpy(d_ciz, ciz, sizeof(ciz), hipMemcpyHostToDevice);

    free(h_pxx);
    free(h_pyy);
    free(h_pzz);
    free(h_pxy);
    free(h_pxz);
    free(h_pyz);

}

