#include "auxFunctions.cuh"
#include "var.cuh"
#include <hip/hip_runtime.h>
#include <fstream>
#include <vector>
#include <iomanip>
#include <string>
#include <cstdlib>
#include <stdexcept>
#include "errorDef.cuh"

#include "precision.cuh"

void freeMemory(dfloat **pointers, int count) {
    for (int i = 0; i < count; ++i) {
        if (pointers[i] != nullptr) {
            hipFree(pointers[i]);
        }
    }
}

void generateSimulationInfoFile(
    const string& filepath, const int nx, const int ny, const int nz, const int stamp, const int nsteps, const dfloat tau, 
    const string& sim_id, const string& fluid_model
) {
    try {
        ofstream file(filepath);

        if (!file.is_open()) {
            cerr << "Erro ao abrir o arquivo: " << filepath << endl;
            return;
        }

        file << "---------------------------- SIMULATION INFORMATION ----------------------------\n"
             << "                           Simulation ID: " << sim_id << '\n'
             << "                           Velocity set: " << fluid_model << '\n'
             << "                           Precision: " << PRECISION_TYPE << '\n'
             << "                           NX: " << nx << '\n'
             << "                           NY: " << ny << '\n'
             << "                           NZ: " << nz << '\n'
             << "                           NZ_TOTAL: " << nz << '\n'
             << "                           Tau: " << tau << '\n'
             << "                           Umax: 0.000000e+00\n"
             << "                           FX: 0.000000e+00\n"
             << "                           FY: 0.000000e+00\n"
             << "                           FZ: 0.000000e+00\n"
             << "                           Save steps: " << stamp << '\n'
             << "                           Nsteps: " << nsteps << '\n'
             << "                           MLUPS: 1.187970e+01\n"
             << "--------------------------------------------------------------------------------\n";

        file.close();
        cout << "Arquivo de informações da simulação criado em: " << filepath << endl;
    } catch (const exception& e) {
        cerr << "Erro ao gerar o arquivo de informações: " << e.what() << endl;
    }
}
