#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "var.cuh"
#include <math.h>

#include "precision.cuh"

#define IDX3D(i,j,k) ((i) + (j) * nx + (k) * nx * ny)
#define IDX4D(i,j,k,l) ((i) + (j) * nx + (k) * nx * ny + (l) * nx * ny * nz)

// ============================================================================================== //

__global__ void initPhase(
    dfloat * __restrict__ phi, 
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat bubble_radius = 20.0 * nx / 150.0;

    dfloat dx = i - nx * 0.5;
    dfloat dy = j - ny * 0.5;
    dfloat dz = k - nz * 0.5;
    dfloat Ri = sqrt((dx * dx) / 4.0 + dy * dy + dz * dz);

    dfloat phi_val = 0.5 + 0.5 * tanh(2.0 * (bubble_radius - Ri) / 3.0);

    phi[idx3D] = phi_val;
}

__global__ void initDist(
    const dfloat * __restrict__ rho, 
    const dfloat * __restrict__ phi, 
    dfloat * __restrict__ f,
    dfloat * __restrict__ g,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat rho_val = rho[idx3D];
    dfloat phi_val = phi[idx3D];

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        f[idx4D] = W[l] * rho_val;
    }

    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        g[idx4D] = W_G[l] * phi_val;
    }
}

// ============================================================================================== //

__global__ void phiCalc(
    dfloat * __restrict__ phi,
    const dfloat * __restrict__ g,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat sum = 0.0;       
    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        sum += g[idx4D];
    }

    phi[idx3D] = sum;
}

__global__ void gradCalc(
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ mod_grad,
    dfloat * __restrict__ normx,
    dfloat * __restrict__ normy,
    dfloat * __restrict__ normz,
    dfloat * __restrict__ indicator,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat grad_fix = 0.0, grad_fiy = 0.0, grad_fiz = 0.0;
    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int ii = i + CIX[l];
        int jj = j + CIY[l];
        int kk = k + CIZ[l];
        int offset = ii + jj * nx + kk * nx * ny;
        dfloat val = phi[offset];
        dfloat coef = 3.0 * W[l];
        grad_fix += coef * CIX[l] * val;
        grad_fiy += coef * CIY[l] * val;
        grad_fiz += coef * CIZ[l] * val;
    }

    dfloat gmag_sq = grad_fix * grad_fix + grad_fiy * grad_fiy + grad_fiz * grad_fiz;
    dfloat gmag = sqrt(gmag_sq);

    mod_grad[idx3D] = gmag;
    normx[idx3D] = grad_fix / (gmag + 1e-9);
    normy[idx3D] = grad_fiy / (gmag + 1e-9);
    normz[idx3D] = grad_fiz / (gmag + 1e-9);
    indicator[idx3D] = gmag;
}

__global__ void curvatureCalc(
    dfloat * __restrict__ curvature,
    const dfloat * __restrict__ indicator,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    dfloat * __restrict__ ffx,
    dfloat * __restrict__ ffy,
    dfloat * __restrict__ ffz,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;

    dfloat normx_ = normx[idx3D];
    dfloat normy_ = normy[idx3D];
    dfloat normz_ = normz[idx3D];
    dfloat ind_ = indicator[idx3D];
    dfloat curv = 0.0;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int ii = i + CIX[l];
        int jj = j + CIY[l];
        int kk = k + CIZ[l];
        int offset = ii + jj * nx + kk * nx * ny;
        dfloat normxN = normx[offset];
        dfloat normyN = normy[offset];
        dfloat normzN = normz[offset];
        dfloat coef = 3.0 * W[l];
        curv -= coef * (CIX[l] * normxN + CIY[l] * normyN + CIZ[l] * normzN);
    }

    dfloat mult = SIGMA * curv * ind_;
    curvature[idx3D] = curv;
    ffx[idx3D] = mult * normx_;
    ffy[idx3D] = mult * normy_;
    ffz[idx3D] = mult * normz_;
}

__global__ void momentiCalc(
    dfloat * __restrict__ ux,
    dfloat * __restrict__ uy,
    dfloat * __restrict__ uz,
    dfloat * __restrict__ rho,
    dfloat * __restrict__ ffx,
    dfloat * __restrict__ ffy,
    dfloat * __restrict__ ffz,
    const dfloat * __restrict__ f,
    dfloat * __restrict__ pxx,
    dfloat * __restrict__ pyy,
    dfloat * __restrict__ pzz,
    dfloat * __restrict__ pxy,
    dfloat * __restrict__ pxz,
    dfloat * __restrict__ pyz,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = i + j * nx + k * nx * ny;
    
    dfloat fneq[FPOINTS];
    dfloat fVal[FPOINTS];

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int idx4D = idx3D + l * nx * ny * nz;
        fVal[l] = f[idx4D];
    }
    
    dfloat rhoOld = rho[idx3D];
    dfloat ffx_val = ffx[idx3D];
    dfloat ffy_val = ffy[idx3D];
    dfloat ffz_val = ffz[idx3D];

    #ifdef FD3Q19
        dfloat sumUx = fVal[1] - fVal[2] + fVal[7] - fVal[8] +
            fVal[9] - fVal[10] + fVal[13] - fVal[14] +
            fVal[15] - fVal[16];
        dfloat sumUy = fVal[3] - fVal[4] + fVal[7] - fVal[8] +
            fVal[11] - fVal[12] - fVal[13] + fVal[14] +
            fVal[17] - fVal[18];
        dfloat sumUz = fVal[5] - fVal[6] + fVal[9] - fVal[10] +
            fVal[11] - fVal[12] - fVal[15] + fVal[16] -
            fVal[17] + fVal[18];
    #elif defined(FD3Q27)
        dfloat sumUx = fVal[1] - fVal[2] + fVal[7] - fVal[8] + fVal[9] - 
            fVal[10] + fVal[13] - fVal[14] + fVal[15] - 
            fVal[16] + fVal[19] - fVal[20] + fVal[21] - 
            fVal[22] + fVal[23] - fVal[24] - fVal[25] + 
            fVal[26];
        dfloat sumUy = fVal[3] - fVal[4] + fVal[7] - fVal[8] + fVal[11] - 
            fVal[12] - fVal[13] + fVal[14] + fVal[17] - 
            fVal[18] + fVal[19] - fVal[20] + fVal[21] - 
            fVal[22] - fVal[23] + fVal[24] + fVal[25] - 
            fVal[26];
        dfloat sumUz = fVal[5] - fVal[6] + fVal[9] - fVal[10] + fVal[11] - 
            fVal[12] - fVal[15] + fVal[16] - fVal[17] + 
            fVal[18] + fVal[19] - fVal[20] - fVal[21] + 
            fVal[22] + fVal[23] - fVal[24] + fVal[25] - 
            fVal[26];
    #endif

    dfloat invRhoOld = 1.0 / rhoOld;
    dfloat halfFx = ffx_val * 0.5 * invRhoOld;
    dfloat halfFy = ffy_val * 0.5 * invRhoOld;
    dfloat halfFz = ffz_val * 0.5 * invRhoOld;

    dfloat uxVal = sumUx * invRhoOld + halfFx;
    dfloat uyVal = sumUy * invRhoOld + halfFy;
    dfloat uzVal = sumUz * invRhoOld + halfFz;

    dfloat rhoNew = 0.0;
    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l)
        rhoNew += fVal[l];
    rho[idx3D] = rhoNew;

    dfloat invCssq = 1.0 / CSSQ;
    dfloat uu = 0.5 * (uxVal * uxVal + uyVal * uyVal + uzVal * uzVal) * invCssq;
    dfloat invRhoNewCssq = 1.0 / (rhoNew * CSSQ);

    dfloat sumXX = 0.0, sumYY = 0.0, sumZZ = 0.0;
    dfloat sumXY = 0.0, sumXZ = 0.0, sumYZ = 0.0;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        dfloat udotc = (uxVal * CIX[l] + uyVal * CIY[l] + uzVal * CIZ[l]) * invCssq;
        dfloat udotc2 = udotc * udotc;
        dfloat eqBase = rhoNew * (udotc + 0.5 * udotc2 - uu);
        dfloat common = W[l] * (rhoNew + eqBase);
        dfloat feq = common;
        dfloat HeF = common * ((CIX[l] - uxVal) * ffx_val +
                               (CIY[l] - uyVal) * ffy_val +
                               (CIZ[l] - uzVal) * ffz_val) * invRhoNewCssq;
        feq -= 0.5 * HeF;
        fneq[l] = fVal[l] - feq;
    }

    #ifdef FD3Q19
        sumXX = fneq[1] + fneq[2] + fneq[7] + fneq[8] +
                fneq[9] + fneq[10] + fneq[13] + fneq[14] +
                fneq[15] + fneq[16];
        sumYY = fneq[3] + fneq[4] + fneq[7] + fneq[8] +
                fneq[11] + fneq[12] + fneq[13] + fneq[14] +
                fneq[17] + fneq[18];
        sumZZ = fneq[5] + fneq[6] + fneq[9] + fneq[10] +
                fneq[11] + fneq[12] + fneq[15] + fneq[16] +
                fneq[17] + fneq[18];
        sumXY = fneq[7] + fneq[8] - fneq[13] - fneq[14];
        sumXZ = fneq[9] + fneq[10] - fneq[15] - fneq[16];
        sumYZ = fneq[11] + fneq[12] - fneq[17] - fneq[18];
    #elif defined(FD3Q27)
        sumXX = fneq[1] + fneq[2] + fneq[7] + fneq[8] + fneq[9] + 
                fneq[10] + fneq[13] + fneq[14] + fneq[15] + fneq[16] + 
                fneq[19] + fneq[20] + fneq[21] + fneq[22] + fneq[23] + 
                fneq[24] + fneq[25] + fneq[26];
        sumYY = fneq[3] + fneq[4] + fneq[7] + fneq[8] + fneq[11] + 
                fneq[12] + fneq[13] + fneq[14] + fneq[17] + fneq[18] + 
                fneq[19] + fneq[20] + fneq[21] + fneq[22] + fneq[23] + 
                fneq[24] + fneq[25] + fneq[26];
        sumZZ = fneq[5] + fneq[6] + fneq[9] + fneq[10] + fneq[11] + 
                fneq[12] + fneq[13] + fneq[14] + fneq[17] + fneq[18] + 
                fneq[19] + fneq[20] + fneq[21] + fneq[22] + fneq[23] + 
                fneq[24] + fneq[25] + fneq[26];
        sumXY = fneq[7] + fneq[8] + fneq[19] + fneq[20] + fneq[21] + 
                fneq[22] - fneq[13] - fneq[14] - fneq[23] - fneq[24] - 
                fneq[25] - fneq[26];
        sumXZ = fneq[9] + fneq[10] + fneq[19] + fneq[20] + fneq[23] + 
                fneq[24] - fneq[15] - fneq[16] - fneq[21] - fneq[22] - 
                fneq[25] - fneq[26];
        sumYZ = fneq[11] + fneq[12] + fneq[19] + fneq[20] + fneq[25] + 
                fneq[26] - fneq[17] - fneq[18] - fneq[21] - fneq[22] - 
                fneq[23] - fneq[24];
    #endif

    pxx[idx3D] = sumXX;
    pyy[idx3D] = sumYY;
    pzz[idx3D] = sumZZ;
    pxy[idx3D] = sumXY;
    pxz[idx3D] = sumXZ;
    pyz[idx3D] = sumYZ;

    ux[idx3D] = uxVal;
    uy[idx3D] = uyVal;
    uz[idx3D] = uzVal;
}

/* GUO ET AL
__global__ void collisionCalc(
    const dfloat * __restrict__ ux,
    const dfloat * __restrict__ uy,
    const dfloat * __restrict__ uz,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    const dfloat * __restrict__ ffx,
    const dfloat * __restrict__ ffy,
    const dfloat * __restrict__ ffz,
    const dfloat * __restrict__ rho,
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ g,
    const dfloat * __restrict__ pxx,
    const dfloat * __restrict__ pyy,
    const dfloat * __restrict__ pzz,
    const dfloat * __restrict__ pxy,
    const dfloat * __restrict__ pxz,
    const dfloat * __restrict__ pyz,
    int nx, int ny, int nz,
    dfloat * __restrict__ f_coll
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1 || k == 0 || k == nz - 1) return;
    
    int idx3D = i + j * nx + k * nx * ny;
    int nxyz  = nx * ny * nz;
    
    dfloat u_x    = ux[idx3D];
    dfloat u_y    = uy[idx3D];
    dfloat u_z    = uz[idx3D];
    dfloat rho_val= rho[idx3D];
    dfloat phi_val= phi[idx3D];
    
    dfloat Fx = ffx[idx3D];
    dfloat Fy = ffy[idx3D];
    dfloat Fz = ffz[idx3D];
    
    dfloat u_sq = u_x*u_x + u_y*u_y + u_z*u_z;
    dfloat uu   = 0.5 * u_sq / CSSQ;
    
    dfloat force_factor = 1.0 - 0.5 * OMEGA;
    
    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        dfloat udotc = (u_x * CIX[l] + u_y * CIY[l] + u_z * CIZ[l]) / CSSQ;
        dfloat feq = W[l] * rho_val * (1.0 + udotc + 0.5 * udotc * udotc - uu);
        
        dfloat ci_dot_u = (u_x * CIX[l] + u_y * CIY[l] + u_z * CIZ[l]);
        dfloat term_x = (CIX[l] - u_x) / CSSQ + (ci_dot_u * CIX[l]) / (CSSQ * CSSQ);
        dfloat term_y = (CIY[l] - u_y) / CSSQ + (ci_dot_u * CIY[l]) / (CSSQ * CSSQ);
        dfloat term_z = (CIZ[l] - u_z) / CSSQ + (ci_dot_u * CIZ[l]) / (CSSQ * CSSQ);
        dfloat Fi = force_factor * W[l] * (term_x * Fx + term_y * Fy + term_z * Fz);
        
        f_coll[idx3D + l * nxyz] = feq + Fi;
    }
    
    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        dfloat udotc = (u_x * CIX[l] + u_y * CIY[l] + u_z * CIZ[l]) / CSSQ;
        dfloat feq_g = W_G[l] * phi_val * (1.0 + udotc);
        dfloat Hi = SHARP_C * phi_val * (1.0 - phi_val) *
                    (CIX[l] * normx[idx3D] + CIY[l] * normy[idx3D] + CIZ[l] * normz[idx3D]);
        g[idx3D + l * nxyz] = feq_g + W_G[l] * Hi;
    }
} 
*/

/* OLD COLLISION KERNEL
__global__ void collisionCalc(
    const dfloat * __restrict__ ux,
    const dfloat * __restrict__ uy,
    const dfloat * __restrict__ uz,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    const dfloat * __restrict__ ffx,
    const dfloat * __restrict__ ffy,
    const dfloat * __restrict__ ffz,
    const dfloat * __restrict__ rho,
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ g,
    const dfloat * __restrict__ pxx,
    const dfloat * __restrict__ pyy,
    const dfloat * __restrict__ pzz,
    const dfloat * __restrict__ pxy,
    const dfloat * __restrict__ pxz,
    const dfloat * __restrict__ pyz,
    int nx, int ny, int nz,
    dfloat * __restrict__ f_coll
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1 || k == 0 || k == nz - 1) return;

    int idx3D = i + j * nx + k * nx * ny;
    int nxyz = nx * ny * nz;

    dfloat ux_val = ux[idx3D];
    dfloat uy_val = uy[idx3D];
    dfloat uz_val = uz[idx3D];
    dfloat rho_val = rho[idx3D];
    dfloat phi_val = phi[idx3D];
    dfloat ffx_val = ffx[idx3D];
    dfloat ffy_val = ffy[idx3D];
    dfloat ffz_val = ffz[idx3D];
    dfloat pxx_val = pxx[idx3D];
    dfloat pyy_val = pyy[idx3D]; 
    dfloat pzz_val = pzz[idx3D];
    dfloat pxy_val = pxy[idx3D];
    dfloat pxz_val = pxz[idx3D];
    dfloat pyz_val = pyz[idx3D];
    dfloat normx_val = normx[idx3D];
    dfloat normy_val = normy[idx3D];
    dfloat normz_val = normz[idx3D];

    dfloat uu = 0.5 * (ux_val * ux_val + uy_val * uy_val + uz_val * uz_val) / CSSQ;
    dfloat one_minus_omega = 1.0 - OMEGA;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) / CSSQ;
        dfloat feq = W[l] * (rho_val + rho_val * (udotc + 0.5 * udotc * udotc - uu));
        dfloat HeF = 0.5 * feq *
                    ((CIX[l] - ux_val) * ffx_val +
                     (CIY[l] - uy_val) * ffy_val +
                     (CIZ[l] - uz_val) * ffz_val) / (rho_val * CSSQ);
        dfloat fneq = (CIX[l] * CIX[l] - CSSQ) * pxx_val +
                      (CIY[l] * CIY[l] - CSSQ) * pyy_val +
                      (CIZ[l] * CIZ[l] - CSSQ) * pzz_val +
                       2.0 * CIX[l] * CIY[l] * pxy_val +
                       2.0 * CIX[l] * CIZ[l] * pxz_val +
                       2.0 * CIY[l] * CIZ[l] * pyz_val;
        f_coll[idx3D + l * nxyz] = feq + one_minus_omega * (W[l] / (2.0 * CSSQ * CSSQ)) * fneq + HeF;
    }
    
    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) / CSSQ;
        dfloat feq = W_G[l] * phi_val * (1 + udotc);
        dfloat Hi = SHARP_C * phi_val * (1 - phi_val) *
                        (CIX[cl] * normx_val + CIY[l] * normy_val + CIZ[l] * normz_val);
        g[idx3D + l * nxyz] = feq + W_G[l] * Hi;
    }
}
*/

__global__ void collisionCalc(
    const dfloat * __restrict__ ux,
    const dfloat * __restrict__ uy,
    const dfloat * __restrict__ uz,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    const dfloat * __restrict__ ffx,
    const dfloat * __restrict__ ffy,
    const dfloat * __restrict__ ffz,
    const dfloat * __restrict__ rho,
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ g,
    const dfloat * __restrict__ pxx,
    const dfloat * __restrict__ pyy,
    const dfloat * __restrict__ pzz,
    const dfloat * __restrict__ pxy,
    const dfloat * __restrict__ pxz,
    const dfloat * __restrict__ pyz,
    int nx, int ny, int nz,
    dfloat * __restrict__ f_coll
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx - 1 || j == 0 || j == ny - 1 || k == 0 || k == nz - 1) return;

    int idx3D = i + j * nx + k * nx * ny;
    int nxyz  = nx * ny * nz;

    dfloat ux_val = ux[idx3D], uy_val = uy[idx3D], uz_val = uz[idx3D];
    dfloat rho_val = rho[idx3D], phi_val = phi[idx3D];
    dfloat ffx_val = ffx[idx3D], ffy_val = ffy[idx3D], ffz_val = ffz[idx3D];
    dfloat pxx_val = pxx[idx3D], pyy_val = pyy[idx3D], pzz_val = pzz[idx3D];
    dfloat pxy_val = pxy[idx3D], pxz_val = pxz[idx3D], pyz_val = pyz[idx3D];
    dfloat normx_val = normx[idx3D], normy_val = normy[idx3D], normz_val = normz[idx3D];

    dfloat u_sq = ux_val*ux_val + uy_val*uy_val + uz_val*uz_val;
    dfloat uu   = 0.5 * u_sq / CSSQ;
    dfloat inv_rho_CSSQ = 1.0 / (rho_val * CSSQ);
    dfloat omc = 1.0 - OMEGA;
    dfloat invCSSQ = 1.0 / CSSQ;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) * invCSSQ;
        dfloat feq = W[l] * rho_val * (1.0 + udotc + 0.5f * udotc * udotc - uu);
        dfloat HeF = 0.5f * feq * ( (CIX[l] - ux_val) * ffx_val +
                                     (CIY[l] - uy_val) * ffy_val +
                                     (CIZ[l] - uz_val) * ffz_val ) * inv_rho_CSSQ;
        dfloat fneq = (CIX[l]*CIX[l] - CSSQ) * pxx_val +
                      (CIY[l]*CIY[l] - CSSQ) * pyy_val +
                      (CIZ[l]*CIZ[l] - CSSQ) * pzz_val +
                      2.0 * CIX[l] * CIY[l] * pxy_val +
                      2.0 * CIX[l] * CIZ[l] * pxz_val +
                      2.0 * CIY[l] * CIZ[l] * pyz_val;
        f_coll[idx3D + l * nxyz] = feq + omc * (W[l] / (2.0f * CSSQ * CSSQ)) * fneq + HeF;
    }

    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) * invCSSQ;
        dfloat feq_g = W_G[l] * phi_val * (1.0 + udotc);
        dfloat Hi = SHARP_C * phi_val * (1.0 - phi_val) *
                    (CIX[l] * normx_val + CIY[l] * normy_val + CIZ[l] * normz_val);
        g[idx3D + l * nxyz] = feq_g + W_G[l] * Hi;
    }
}

__global__ void streamingCalcNew(
    const dfloat * __restrict__ f_coll,
    int nx, int ny, int nz,
    dfloat * __restrict__ f 
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int NxNy = nx * ny;
    int NxNyNz = NxNy * nz;
    int dstBase = i + j * nx + k * NxNy;

    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int src_i = (i - CIX[l] + nx) & (nx-1); // & (nx-1) if div by 2
        int src_j = (j - CIY[l] + ny) & (ny-1);
        int src_k = (k - CIZ[l] + nz) & (nz-1);
        int srcBase = src_i + src_j * nx + src_k * NxNy;
        int dstIdx = l * NxNyNz + dstBase;
        int srcIdx = l * NxNyNz + srcBase;
        f[dstIdx] = f_coll[srcIdx];
    }
}

__global__ void streamingCalc(
    const dfloat * __restrict__ g_in,
    dfloat * __restrict__ g_out,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int NxNy = nx * ny;
    int NxNyNz = NxNy * nz;
    int dstBase = i + j * nx + k * NxNy;

    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        int src_i = (i - CIX[l] + nx) & (nx-1); // & (nx-1) if div by 2
        int src_j = (j - CIY[l] + ny) & (ny-1);
        int src_k = (k - CIZ[l] + nz) & (nz-1);
        int srcBase = src_i + src_j * nx + src_k * NxNy;
        int dstIdx = l * NxNyNz + dstBase;
        int srcIdx = l * NxNyNz + srcBase;
        g_out[dstIdx] = g_in[srcIdx];
    }
}

__global__ void fgBoundary_f(
    dfloat * __restrict__ f,
    const dfloat * __restrict__ rho,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if(i >= nx || j >= ny || k >= nz) return;
    
    #pragma unroll 19
    for (int l = 0; l < FPOINTS; ++l) {
        int bi = i - CIX[l];
        int bj = j - CIY[l];
        int bk = k - CIZ[l];
        if(bi < 0 || bi >= nx || bj < 0 || bj >= ny || bk < 0 || bk >= nz)
            continue;
        if(bi == 0 || bi == nx-1 || bj == 0 || bj == ny-1 || bk == 0 || bk == nz-1) {
            int boundary_idx = bi + bj * nx + bk * nx * ny;
            f[IDX4D(i,j,k,l)] = rho[boundary_idx] * W[l];
        }
    }
}

__global__ void fgBoundary_g(
    dfloat * __restrict__ g,
    const dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if(i >= nx || j >= ny || k >= nz) return;
    
    #pragma unroll 15
    for (int l = 0; l < GPOINTS; ++l) {
        int bi = i - CIX[l];
        int bj = j - CIY[l];
        int bk = k - CIZ[l];
        if(bi < 0 || bi >= nx || bj < 0 || bj >= ny || bk < 0 || bk >= nz)
            continue;
        if(bi == 0 || bi == nx-1 || bj == 0 || bj == ny-1 || bk == 0 || bk == nz-1) {
            int boundary_idx = bi + bj * nx + bk * nx * ny;
            g[IDX4D(i,j,k,l)] = phi[boundary_idx] * W_G[l];
        }
    }
}

__global__ void boundaryConditions_z(
    dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        phi[IDX3D(i,j,0)] = phi[IDX3D(i,j,1)];
        phi[IDX3D(i,j,nz-1)] = phi[IDX3D(i,j,nz-2)];
    }
}

__global__ void boundaryConditions_y(
    dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && k < nz) {
        phi[IDX3D(i,0,k)] = phi[IDX3D(i,1,k)];
        phi[IDX3D(i,ny-1,k)] = phi[IDX3D(i,ny-2,k)];
    }
}

__global__ void boundaryConditions_x(
    dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;

    if (j < ny && k < nz) {
        phi[IDX3D(0,j,k)] = phi[IDX3D(1,j,k)];
        phi[IDX3D(nx-1,j,k)] = phi[IDX3D(nx-2,j,k)];
    }
}