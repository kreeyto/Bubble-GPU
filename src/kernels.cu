#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include "var.cuh"
#include <math.h>

#include "precision.cuh"

#define RHO_BEFORE

#ifdef ROW_MAJOR

    #define IDX3D(i,j,k) ((i) + (j) * nx + (k) * nx * ny)
    #define IDX4D(i,j,k,l) ((i) + (j) * nx + (k) * nx * ny + (l) * nx * ny * nz)
    __device__ __forceinline__ int inline3D(int i, int j, int k, int nx, int ny) {
        return i + j * nx + k * nx * ny;
    }
    __device__ __forceinline__ int inline4D(int i, int j, int k, int l, int nx, int ny, int nz) {
        return inline3D(i,j,k,nx,ny) + l * nx * ny * nz;
    }

#elif defined(COLUMN_MAJOR)

    #define IDX3D(i,j,k) ((j) + (i) * ny + (k) * nx * ny)
    #define IDX4D(i,j,k,l) ((j) + (i) * ny + (k) * nx * ny + (l) * nx * ny * nz)
    __device__ __forceinline__ int inline3D(int i, int j, int k, int nx, int ny) {
        return j + i * ny + k * nx * ny;
    }
    __device__ __forceinline__ int inline4D(int i, int j, int k, int l, int nx, int ny, int nz) {
        return l * nx * ny * nz + inline3D(i,j,k,nx,ny);
    }

#endif

// ============================================================================================== //

__global__ void initTensor(
    dfloat * __restrict__ pxx,
    dfloat * __restrict__ pyy,
    dfloat * __restrict__ pzz,
    dfloat * __restrict__ pxy,
    dfloat * __restrict__ pxz,
    dfloat * __restrict__ pyz,
    dfloat * __restrict__ rho,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    dfloat val = 1.0;
    pxx[idx3D] = val;
    pyy[idx3D] = val;
    pzz[idx3D] = val;
    pxy[idx3D] = val;
    pxz[idx3D] = val;
    pyz[idx3D] = val;
    rho[idx3D] = val;
}

__global__ void initPhase(
    dfloat * __restrict__ phi, 
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    dfloat bubble_radius = 20.0 * nx / 150.0;

    dfloat dx = i - nx * 0.5;
    dfloat dy = j - ny * 0.5;
    dfloat dz = k - nz * 0.5;
    dfloat Ri = sqrt((dx * dx) / 4.0 + dy * dy + dz * dz);

    dfloat phi_val = 0.5 + 0.5 * tanh(2.0 * (bubble_radius - Ri) / 3.0);

    phi[idx3D] = phi_val;
}

// =================================================================================================== //

__global__ void initDist(
    const dfloat * __restrict__ rho, 
    const dfloat * __restrict__ phi, 
    dfloat * __restrict__ f,
    dfloat * __restrict__ g,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    dfloat rho_val = rho[idx3D];
    dfloat phi_val = phi[idx3D];

    for (int l = 0; l < FPOINTS; ++l) {
        int idx4D = inline4D(i,j,k,l,nx,ny,nz);
        f[idx4D] = (W[l] * rho_val) - W[l];
    }

    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = inline4D(i,j,k,l,nx,ny,nz);
        g[idx4D] = W_G[l] * phi_val;
    }
}

// =================================================================================================== //

// THE KERNELS BELOW ARE CALLED ON A LOOP

// ============================================================================================== //

__global__ void phiCalc(
    dfloat * __restrict__ phi,
    const dfloat * __restrict__ g,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    dfloat sum = 0.0;       
    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = inline4D(i,j,k,l,nx,ny,nz);
        sum += g[idx4D];
    }

    phi[idx3D] = sum;
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void gradCalc(
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ normx,
    dfloat * __restrict__ normy,
    dfloat * __restrict__ normz,
    dfloat * __restrict__ indicator,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    dfloat grad_fix = 0.0, grad_fiy = 0.0, grad_fiz = 0.0;
    for (int l = 0; l < FPOINTS; ++l) {
        int ii = i + CIX[l];
        int jj = j + CIY[l];
        int kk = k + CIZ[l];

        int offset = inline3D(ii,jj,kk,nx,ny);
        dfloat val = phi[offset];
        dfloat coef = 3.0 * W[l];
        grad_fix += coef * CIX[l] * val;
        grad_fiy += coef * CIY[l] * val;
        grad_fiz += coef * CIZ[l] * val;
    }

    dfloat gmag_sq = grad_fix * grad_fix + grad_fiy * grad_fiy + grad_fiz * grad_fiz;
    dfloat factor = rsqrtf(fmaxf(gmag_sq, 1e-9));

    normx[idx3D] = grad_fix * factor;
    normy[idx3D] = grad_fiy * factor;
    normz[idx3D] = grad_fiz * factor;
    indicator[idx3D] = gmag_sq * factor;  
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void curvatureCalc(
    dfloat * __restrict__ curvature,
    const dfloat * __restrict__ indicator,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    dfloat * __restrict__ ffx,
    dfloat * __restrict__ ffy,
    dfloat * __restrict__ ffz,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    dfloat normx_ = normx[idx3D];
    dfloat normy_ = normy[idx3D];
    dfloat normz_ = normz[idx3D];
    dfloat ind_ = indicator[idx3D];
    dfloat curv = 0.0;

    for (int l = 0; l < FPOINTS; ++l) {
        int ii = i + CIX[l];
        int jj = j + CIY[l];
        int kk = k + CIZ[l];

        int offset = inline3D(ii,jj,kk,nx,ny);
        dfloat normxN = normx[offset];
        dfloat normyN = normy[offset];
        dfloat normzN = normz[offset];
        dfloat coef = 3.0 * W[l];
        curv -= coef * (CIX[l] * normxN + CIY[l] * normyN + CIZ[l] * normzN);
    }

    dfloat mult = SIGMA * curv;

    curvature[idx3D] = curv;
    ffx[idx3D] = mult * normx_ * ind_;
    ffy[idx3D] = mult * normy_ * ind_;
    ffz[idx3D] = mult * normz_ * ind_;
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void momentiCalc(
    dfloat * __restrict__ ux,
    dfloat * __restrict__ uy,
    dfloat * __restrict__ uz,
    dfloat * __restrict__ rho,
    dfloat * __restrict__ ffx,
    dfloat * __restrict__ ffy,
    dfloat * __restrict__ ffz,
    const dfloat * __restrict__ f,
    dfloat * __restrict__ pxx,
    dfloat * __restrict__ pyy,
    dfloat * __restrict__ pzz,
    dfloat * __restrict__ pxy,
    dfloat * __restrict__ pxz,
    dfloat * __restrict__ pyz,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = inline3D(i,j,k,nx,ny);
    
    dfloat fneq[FPOINTS];
    dfloat fVal[FPOINTS];

    for (int l = 0; l < FPOINTS; ++l) {
        int idx4D = inline4D(i,j,k,l,nx,ny,nz);
        fVal[l] = f[idx4D];
    }

    #ifdef RHO_BEFORE

        dfloat rhoVal = 0.0;
        dfloat rhoShift = 0.0;
        for (int l = 0; l < FPOINTS; ++l)
            rhoShift += fVal[l];
        rhoVal = rhoShift + 1.0;

        dfloat invRho = 1.0 / rhoVal;

        dfloat sumUx = invRho * (fVal[1] - fVal[2] + fVal[7] - fVal[8] + fVal[9] - fVal[10] + fVal[13] - fVal[14] + fVal[15] - fVal[16]);
        dfloat sumUy = invRho * (fVal[3] - fVal[4] + fVal[7] - fVal[8] + fVal[11] - fVal[12] + fVal[14] - fVal[13] + fVal[17] - fVal[18]);
        dfloat sumUz = invRho * (fVal[5] - fVal[6] + fVal[9] - fVal[10] + fVal[11] - fVal[12] + fVal[16] - fVal[15] + fVal[18] - fVal[17]);

    #elif defined(RHO_AFTER)

        dfloat rhoOld = rho[idx3D];
        dfloat invRhoOld = 1.0 / rhoOld;

        dfloat sumUx = invRhoOld * (fVal[1] - fVal[2] + fVal[7] - fVal[8] + fVal[9] - fVal[10] + fVal[13] - fVal[14] + fVal[15] - fVal[16]);
        dfloat sumUy = invRhoOld * (fVal[3] - fVal[4] + fVal[7] - fVal[8] + fVal[11] - fVal[12] + fVal[14] - fVal[13] + fVal[17] - fVal[18]);
        dfloat sumUz = invRhoOld * (fVal[5] - fVal[6] + fVal[9] - fVal[10] + fVal[11] - fVal[12] + fVal[16] - fVal[15] + fVal[18] - fVal[17]);

        dfloat rhoVal = 0.0;
        dfloat rhoShift = 0.0;
        for (int l = 0; l < FPOINTS; ++l)
            rhoShift += fVal[l];
        rhoVal = rhoShift + 1.0;

    #endif

    dfloat ffx_val = ffx[idx3D];
    dfloat ffy_val = ffy[idx3D];
    dfloat ffz_val = ffz[idx3D];

    dfloat halfFx = ffx_val * 0.5;
    dfloat halfFy = ffy_val * 0.5;
    dfloat halfFz = ffz_val * 0.5;

    dfloat uxVal = sumUx + halfFx;
    dfloat uyVal = sumUy + halfFy;
    dfloat uzVal = sumUz + halfFz;

    dfloat invCssq = 1.0 / CSSQ;
    dfloat uu = 0.5 * (uxVal * uxVal + uyVal * uyVal + uzVal * uzVal) * invCssq;
    dfloat invRhoCssq = 1.0 / (rhoVal * CSSQ);

    dfloat sumXX = 0.0, sumYY = 0.0, sumZZ = 0.0;
    dfloat sumXY = 0.0, sumXZ = 0.0, sumYZ = 0.0;

    for (int l = 0; l < FPOINTS; ++l) {
        dfloat udotc = (uxVal * CIX[l] + uyVal * CIY[l] + uzVal * CIZ[l]) * invCssq;
        dfloat udotc2 = udotc * udotc;
        dfloat eqBase = rhoVal * (udotc + 0.5 * udotc2 - uu);
        dfloat common = W[l] * (rhoVal + eqBase);
        dfloat HeF = common * ((CIX[l] - uxVal) * ffx_val +
                               (CIY[l] - uyVal) * ffy_val +
                               (CIZ[l] - uzVal) * ffz_val) * invRhoCssq;
        dfloat feq = common - 0.5 * HeF; 
        dfloat feq_shifted = feq - W[l];
        fneq[l] = fVal[l] - feq_shifted;
    }

    sumXX = fneq[1] + fneq[2] + fneq[7] + fneq[8] + fneq[9] + fneq[10] + fneq[13] + fneq[14] + fneq[15] + fneq[16];
    sumYY = fneq[3] + fneq[4] + fneq[7] + fneq[8] + fneq[11] + fneq[12] + fneq[13] + fneq[14] + fneq[17] + fneq[18];
    sumZZ = fneq[5] + fneq[6] + fneq[9] + fneq[10] + fneq[11] + fneq[12] + fneq[15] + fneq[16] + fneq[17] + fneq[18];
    sumXY = fneq[7] - fneq[13] + fneq[8] - fneq[14];
    sumXZ = fneq[9] - fneq[15] + fneq[10] - fneq[16];
    sumYZ = fneq[11] - fneq[17] + fneq[12] - fneq[18];

    pxx[idx3D] = sumXX; pyy[idx3D] = sumYY; pzz[idx3D] = sumZZ;
    pxy[idx3D] = sumXY; pxz[idx3D] = sumXZ; pyz[idx3D] = sumYZ;

    ux[idx3D] = uxVal; uy[idx3D] = uyVal; uz[idx3D] = uzVal;
    rho[idx3D] = rhoVal;
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void collisionCalc(
    const dfloat * __restrict__ ux,
    const dfloat * __restrict__ uy,
    const dfloat * __restrict__ uz,
    const dfloat * __restrict__ normx,
    const dfloat * __restrict__ normy,
    const dfloat * __restrict__ normz,
    const dfloat * __restrict__ ffx,
    const dfloat * __restrict__ ffy,
    const dfloat * __restrict__ ffz,
    const dfloat * __restrict__ rho,
    const dfloat * __restrict__ phi,
    dfloat * __restrict__ g,
    const dfloat * __restrict__ pxx,
    const dfloat * __restrict__ pyy,
    const dfloat * __restrict__ pzz,
    const dfloat * __restrict__ pxy,
    const dfloat * __restrict__ pxz,
    const dfloat * __restrict__ pyz,
    int nx, int ny, int nz,
    dfloat * __restrict__ f
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;
    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    dfloat ux_val = ux[idx3D], uy_val = uy[idx3D], uz_val = uz[idx3D];
    dfloat rho_val = rho[idx3D], phi_val = phi[idx3D];
    dfloat ffx_val = ffx[idx3D], ffy_val = ffy[idx3D], ffz_val = ffz[idx3D];
    dfloat pxx_val = pxx[idx3D], pyy_val = pyy[idx3D], pzz_val = pzz[idx3D];
    dfloat pxy_val = pxy[idx3D], pxz_val = pxz[idx3D], pyz_val = pyz[idx3D];
    dfloat normx_val = normx[idx3D], normy_val = normy[idx3D], normz_val = normz[idx3D];

    dfloat u_sq = ux_val*ux_val + uy_val*uy_val + uz_val*uz_val;
    dfloat uu = 0.5 * u_sq / CSSQ;
    dfloat inv_rho_CSSQ = 1.0 / (rho_val * CSSQ);
    dfloat omc = 1.0 - OMEGA;
    dfloat invCSSQ = 1.0 / CSSQ;

    for (int l = 0; l < FPOINTS; ++l) {
        int ii = i + CIX[l];
        int jj = j + CIY[l];
        int kk = k + CIZ[l];

        int offset = inline4D(ii,jj,kk,l,nx,ny,nz);
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) * invCSSQ;
        dfloat feq = W[l] * (rho_val + rho_val * (udotc + 0.5 * udotc*udotc - uu));
        dfloat feq_shifted = feq - W[l];
        dfloat HeF = 0.5 * feq_shifted * ( (CIX[l] - ux_val) * ffx_val +
                                (CIY[l] - uy_val) * ffy_val +
                                (CIZ[l] - uz_val) * ffz_val ) * inv_rho_CSSQ;
        dfloat fneq = (W[l] / (2.0 * CSSQ * CSSQ)) * ((CIX[l]*CIX[l] - CSSQ) * pxx_val +
                                                      (CIY[l]*CIY[l] - CSSQ) * pyy_val +
                                                      (CIZ[l]*CIZ[l] - CSSQ) * pzz_val +
                                                       2.0 * CIX[l] * CIY[l] * pxy_val +
                                                       2.0 * CIX[l] * CIZ[l] * pxz_val +
                                                       2.0 * CIY[l] * CIZ[l] * pyz_val
                                                    );
        f[offset] = feq_shifted + omc * fneq + HeF; 
    }

    dfloat phi_norm = SHARP_C * phi_val * (1.0 - phi_val);
    for (int l = 0; l < GPOINTS; ++l) {
        int idx4D = inline4D(i,j,k,l,nx,ny,nz);
        dfloat udotc = (ux_val * CIX[l] + uy_val * CIY[l] + uz_val * CIZ[l]) * invCSSQ;
        dfloat geq = W_G[l] * phi_val * (1.0 + udotc);
        dfloat Hi = phi_norm * (CIX[l] * normx_val + CIY[l] * normy_val + CIZ[l] * normz_val);
        g[idx4D] = geq + W_G[l] * Hi; // + (1 - omega) * gneq;
        // there is an option to stream g in collision as f is being done
    }
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void streamingCalc(
    const dfloat * __restrict__ g,
    dfloat * __restrict__ g_out,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    for (int l = 0; l < GPOINTS; ++l) {
        int src_i = (i - CIX[l] + nx) & (nx-1);
        int src_j = (j - CIY[l] + ny) & (ny-1);
        int src_k = (k - CIZ[l] + nz) & (nz-1);
        int dstIdx = inline4D(i,j,k,l,nx,ny,nz);
        int srcIdx = inline4D(src_i,src_j,src_k,l,nx,ny,nz);
        g_out[dstIdx] = g[srcIdx];
    }
}

// =================================================================================================== //



// =================================================================================================== //

__global__ void fgBoundary(
    dfloat * __restrict__ f,
    const dfloat * __restrict__ rho,
    dfloat * __restrict__ g,
    const dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= nx || j >= ny || k >= nz) return;

    int idx3D = inline3D(i,j,k,nx,ny);

    if (i == 0 || i == nx-1 || j == 0 || j == ny-1 || k == 0 || k == nz-1) {
        for (int l = 0; l < FPOINTS; ++l) {
            int ni = i + CIX[l];
            int nj = j + CIY[l];
            int nk = k + CIZ[l];
            if (ni >= 0 && ni < nx && nj >= 0 && nj < ny && nk >= 0 && nk < nz) {
                int idx4D = inline4D(ni,nj,nk,l,nx,ny,nz);
                f[idx4D] = (rho[idx3D] - 1.0) * W[l];
            }
        }
        for (int l = 0; l < GPOINTS; ++l) {
            int ni = i + CIX[l];
            int nj = j + CIY[l];
            int nk = k + CIZ[l];
            if (ni >= 0 && ni < nx && nj >= 0 && nj < ny && nk >= 0 && nk < nz) {
                int idx4D = inline4D(ni,nj,nk,l,nx,ny,nz);
                g[idx4D] = phi[idx3D] * W_G[l];
            }
        }
    }
}

__global__ void boundaryConditions(
    dfloat * __restrict__ phi,
    int nx, int ny, int nz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < nx && j < ny) {
        phi[IDX3D(i,j,0)] = phi[IDX3D(i,j,1)];
        phi[IDX3D(i,j,nz-1)] = phi[IDX3D(i,j,nz-2)];
    }

    if (i < nx && k < nz) {
        phi[IDX3D(i,0,k)] = phi[IDX3D(i,1,k)];
        phi[IDX3D(i,ny-1,k)] = phi[IDX3D(i,ny-2,k)];
    }

    if (j < ny && k < nz) {
        phi[IDX3D(0,j,k)] = phi[IDX3D(1,j,k)];
        phi[IDX3D(nx-1,j,k)] = phi[IDX3D(nx-2,j,k)];
    }
}


// =================================================================================================== //

